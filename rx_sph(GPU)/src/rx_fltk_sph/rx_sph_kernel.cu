#include "hip/hip_runtime.h"
/*! 
  @file rx_sph_kernel.cu
	
  @brief CUDA�ɂ��SPH
 
  @author Makoto Fujisawa
  @date 2009-08, 2011-06
*/
// FILE --rx_sph_kernel.cu--

#ifndef _RX_CUSPH_KERNEL_CU_
#define _RX_CUSPH_KERNEL_CU_


//-----------------------------------------------------------------------------
// �C���N���[�h�t�@�C��
//-----------------------------------------------------------------------------
#include "rx_cu_common.cu"

//-----------------------------------------------------------------------------
// �ϐ�
//-----------------------------------------------------------------------------
#if USE_TEX
texture<float4, hipTextureType1D, hipReadModeElementType> dSortedPosTex;
texture<float4, hipTextureType1D, hipReadModeElementType> dSortedVelTex;
texture<uint,   hipTextureType1D, hipReadModeElementType> dCellStartTex;
texture<uint,   hipTextureType1D, hipReadModeElementType> dCellEndTex;
#endif

// �V�~�����[�V�����p�����[�^(�R���X�^���g������)
__constant__ rxSimParams params;

__constant__ int RXNA[] = {0, -1, 1, -2, 2, -3, 3, -4, 4, -5, 5, -6, 6, -7, 7, -8, 8};






//-----------------------------------------------------------------------------
// �O���b�h
//-----------------------------------------------------------------------------
/*!
 * �O���b�h�ʒu�v�Z
 * @param[in] p ���W
 * @return �O���b�h���W
 */
__device__ 
int3 calcGridPos(float3 p)
{
	int3 gridPos;
	gridPos.x = floor((p.x-params.WorldOrigin.x)/params.CellWidth.x);
	gridPos.y = floor((p.y-params.WorldOrigin.y)/params.CellWidth.y);
	gridPos.z = floor((p.z-params.WorldOrigin.z)/params.CellWidth.z);

	gridPos.x = min(max(gridPos.x, 0), params.GridSize.x-1);
	gridPos.y = min(max(gridPos.y, 0), params.GridSize.y-1);
	gridPos.z = min(max(gridPos.z, 0), params.GridSize.z-1);

	return gridPos;
}

/*!
 * �O���b�h���W����1�����z�񒆂ł̈ʒu���v�Z
 * @param[in] gridPos �O���b�h���W
 * @return �A�h���X
 */
__device__ 
uint calcGridHash(int3 gridPos)
{
	return __umul24(__umul24(gridPos.z, params.GridSize.y), params.GridSize.x)+__umul24(gridPos.y, params.GridSize.x)+gridPos.x;
}



//-----------------------------------------------------------------------------
// �n�b�V��
//-----------------------------------------------------------------------------
/*!
 * �e�p�[�e�B�N���̃O���b�h�n�b�V���l
 * @param[out] gridParticleHash
 * @param[out] dSortedIndex
 * @param[in] pos �p�[�e�B�N���ʒu���i�[�����z��
 * @param[in] nprts �p�[�e�B�N����
 */
__global__
void calcHashD(uint*   dGridParticleHash, 
			   uint*   dSortedIndex, 
			   float4* dPos, 
			   uint	   nprts)
{
	uint index = __umul24(blockIdx.x, blockDim.x)+threadIdx.x;
	if(index >= nprts) return;
	
	volatile float4 p = dPos[index];

	int3 gridPos = calcGridPos(make_float3(p.x, p.y, p.z));
	uint hash = calcGridHash(gridPos);

	dGridParticleHash[index] = hash;
	dSortedIndex[index] = index;
}

/*!
 * �p�[�e�B�N���f�[�^���\�[�g���āC�n�b�V�����̊e�Z���̍ŏ��̃A�h���X������
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] oldPos �p�[�e�B�N���ʒu
 * @param[in] oldVel �p�[�e�B�N�����x
 */
__global__
void reorderDataAndFindCellStartD(rxParticleCell cell, float4* dSortedPos, float4* dSortedVel)
{
	extern __shared__ uint sharedHash[];	// �T�C�Y : blockSize+1
	uint index = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
	
	uint hash;
	if(index < cell.uNumParticles){
		hash = cell.dGridParticleHash[index];	// �n�b�V���l

		sharedHash[threadIdx.x+1] = hash;	// �n�b�V���l���V�F�A�[�h�������Ɋi�[

		if(index > 0 && threadIdx.x == 0){
			// �e�V�F�A�[�h�������̍ŏ��ׂ͗̃O���b�h�̃p�[�e�B�N���̃n�b�V���l���i�[
			sharedHash[0] = cell.dGridParticleHash[index-1];
		}
	}

	__syncthreads();
	
	if(index < cell.uNumParticles){
		// �C���f�b�N�X0�ł���C�������́C��O�̃p�[�e�B�N���̃O���b�h�n�b�V���l���قȂ�ꍇ�C
		// �p�[�e�B�N���͕����̈�̍ŏ�
		if(index == 0 || hash != sharedHash[threadIdx.x]){
			cell.dCellStart[hash] = index;
			if(index > 0){
				// ��O�̃p�[�e�B�N���́C��O�̕����̈�̍Ō�
				cell.dCellEnd[sharedHash[threadIdx.x]] = index;
			}
		}

		// �C���f�b�N�X���Ō�Ȃ�΁C�����̈�̍Ō�
		if(index == cell.uNumParticles-1){
			cell.dCellEnd[hash] = index+1;
		}

		// �ʒu�Ƒ��x�̃f�[�^����ёւ�
		// �\�[�g�����C���f�b�N�X�ŎQ�Ƃ��\�����T�����̃O���[�o���������A�N�Z�X���ɗ͗}���邽�߂Ƀf�[�^���̂��̂���ёւ���
		uint sortedIndex = cell.dSortedIndex[index];
		float4 pos = FETCH(dSortedPos, sortedIndex);
		float4 vel = FETCH(dSortedVel, sortedIndex);

		cell.dSortedPos[index] = pos;
		cell.dSortedVel[index] = vel;
	}
}




//-----------------------------------------------------------------------------
// MARK:SPH
//-----------------------------------------------------------------------------
/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋������疧�x���v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] index �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float calDensityCell(int3 gridPos, uint i, float3 pos0, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float h = params.EffectiveRadius;
	float dens = 0.0f;
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);
		for(uint j = startIndex; j < endIndex; ++j){
			//if(j == i) continue;

			float3 pos1 = make_float3(FETCHC(dSortedPos, j));

			float3 rij = pos0-pos1;
			float r = length(rij);

			if(r <= h){
				float q = h*h-r*r;
				dens += params.Mass*params.Wpoly6*q*q*q;
			}
		}
	}

	return dens;
}

/*!
 * �p�[�e�B�N�����x�v�Z(�J�[�l���֐�)
 * @param[out] newDens �p�[�e�B�N�����x
 * @param[out] newPres �p�[�e�B�N������
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void sphCalDensity(float* newDens, float* newPres, rxParticleCell cell)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	float3 pos = make_float3(FETCHC(dSortedPos, index));	// �p�[�e�B�N���ʒu
	//int3 grid_pos = calcGridPos(pos);	// �p�[�e�B�N����������O���b�h�ʒu
	float h = params.EffectiveRadius;

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos-make_float3(h));
	grid_pos1 = calcGridPos(pos+make_float3(h));

	// ���͂̃O���b�h���܂߂ċߖT�T���C���x�v�Z
	float dens = 0.0f;
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				dens += calDensityCell(n_grid_pos, index, pos, cell);
			}
		}
	}

	// �K�X�萔���g�������͎Z�o
	float pres;
	pres = params.GasStiffness*(dens-params.Density);

	// ���x�ƈ��͒l�����ʂɏ�������
	uint oIdx = cell.dSortedIndex[index];
	newDens[oIdx] = dens;
	newPres[oIdx] = pres;
}




/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋�������@�����v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] i �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float3 calNormalCell(int3 gridPos, uint i, float3 pos0, float* dens, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float h = params.EffectiveRadius;
	float3 nrm = make_float3(0.0f);
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);

		for(uint j = startIndex; j < endIndex; ++j){
			if(j != i){
				float3 pos1 = make_float3(FETCHC(dSortedPos, j));

				float3 rij = pos0-pos1;
				float r = length(rij);

				if(r <= h && r > 0.0001){
					float d1 = dens[cell.dSortedIndex[j]];
					float q = h*h-r*r;

					nrm += (params.Mass/d1)*params.GWpoly6*q*q*rij;
				}

			}
		}
	}

	return nrm;
}


/*!
 * �p�[�e�B�N���@���v�Z(�J�[�l���֐�)
 * @param[out] newNrms �p�[�e�B�N���@��
 * @param[in] dens �p�[�e�B�N�����x
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void sphCalNormal(float4* newNrms, float* dens, rxParticleCell cell)
{
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	float3 pos = make_float3(FETCHC(dSortedPos, index));	// �p�[�e�B�N���ʒu
	float h = params.EffectiveRadius;
	//int3 grid_pos = calcGridPos(pos);	// �p�[�e�B�N����������O���b�h�ʒu

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos-make_float3(h));
	grid_pos1 = calcGridPos(pos+make_float3(h));

	// ���͂̃O���b�h���܂߂ċߖT�T���C���x�v�Z
	float3 nrm = make_float3(0.0f);
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				nrm += calNormalCell(n_grid_pos, index, pos, dens, cell);
			}
		}
	}

	float l = length(nrm);
	if(l > 0){
		nrm /= l;
	}

	uint oIdx = cell.dSortedIndex[index];
	newNrms[oIdx] = make_float4(nrm, 0.0f);
}




/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋�������͏���v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] i �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos0 �v�Z���W
 * @param[in] vel0 �v�Z���W�̑��x
 * @param[in] dens0 �v�Z���W�̖��x
 * @param[in] pres0 �v�Z���W�̈���
 * @param[in] dens �p�[�e�B�N�����x
 * @param[in] pres �p�[�e�B�N������
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�����͏�
 */
__device__
float3 calForceCell(int3 gridPos, uint i, float3 pos0, float3 vel0, float dens0, float pres0, 
					float* dens, float* pres, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float h = params.EffectiveRadius;

	float3 frc = make_float3(0.0f);
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);
		float prsi = pres0/(dens0*dens0);
		for(uint j = startIndex; j < endIndex; ++j){
			if(j != i){
				// �ߖT�p�[�e�B�N���̃p�����[�^
				float3 pos1 = make_float3(FETCHC(dSortedPos, j));
				float3 vel1 = make_float3(FETCHC(dSortedVel, j));

				float3 rij = pos0-pos1;
				float r = length(rij);

				if(r <= h && r > 0.0001){
					//float3 vel1 = make_float3(vel[cell.dSortedIndex[j]]);
					float dens1 = dens[cell.dSortedIndex[j]];
					float pres1 = pres[cell.dSortedIndex[j]];

					float3 vji = vel1-vel0;

					float prsj = pres1/(dens1*dens1);
					float q = h-r;

					// ���͍�
					frc += -dens0*params.Mass*(prsi+prsj)*params.GWspiky*q*q*rij/r;

					// �S����
					frc += params.Viscosity*params.Mass*(vji/dens1)*params.LWvisc*q;
				}
			}
		}
	}

	return frc;
}

/*!
 * �p�[�e�B�N���ɂ�����͂̌v�Z(�J�[�l���֐�)
 * @param[in] dens �p�[�e�B�N�����x
 * @param[in] pres �p�[�e�B�N������
 * @param[out] outFrc �p�[�e�B�N���ɂ������
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void sphCalForces(float* dens, float* pres, float4* outFrc, rxParticleCell cell)
{
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	// �\�[�g�ςݔz�񂩂�p�[�e�B�N���f�[�^���擾
	float3 pos0 = make_float3(FETCHC(dSortedPos, index));
	float3 vel0 = make_float3(FETCHC(dSortedVel, index));

	int3 gridPos0 = calcGridPos(pos0);

	// �p�[�e�B�N���̃\�[�g�Ȃ��z���ł̃C���f�b�N�X
	uint oIdx = cell.dSortedIndex[index];

	float dens0 = dens[oIdx];
	float pres0 = pres[oIdx];

	float h = params.EffectiveRadius;
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos0-make_float3(h));
	grid_pos1 = calcGridPos(pos0+make_float3(h));

	// ���͂̃O���b�h���܂߂ċߖT�T���C���͍��C�S�������v�Z
	float3 frc = make_float3(0.0f);
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);

				frc += calForceCell(n_grid_pos, index, pos0, vel0, dens0, pres0, dens, pres, cell);
			}
		}
	}

	// �O��(�d�͂╂�͂Ȃ�)
	frc += params.Gravity*dens0;

	outFrc[oIdx] = make_float4(frc, 0.0f);
}


__device__
void calCollisionSolid(float3 &pos, float3 &vel, float dt)
{
	float d;
	float3 n;
	float3 cp;

	// �{�b�N�X�`��̃I�u�W�F�N�g�Ƃ̏Փ�
#if MAX_BOX_NUM
	for(int i = 0; i < params.BoxNum; ++i){
		if(params.BoxFlg[i] == 0) continue;
		
		collisionPointBox(pos, params.BoxCen[i], params.BoxExt[i], params.BoxRot[i], params.BoxInvRot[i], cp, d, n);

		if(d < 0.0){
			float res = params.Restitution;
			res = (res > 0) ? (res*fabs(d)/(dt*length(vel))) : 0.0f;
			vel -= (1+res)*n*dot(n, vel);
			pos = cp;
		}
	}
#endif

	// ���`��̃I�u�W�F�N�g�Ƃ̏Փ�
#if MAX_SPHERE_NUM
	for(int i = 0; i < params.SphereNum; ++i){
		if(params.SphereFlg[i] == 0) continue;

		collisionPointSphere(pos, params.SphereCen[i], params.SphereRad[i], cp, d, n);

		if(d < 0.0){
			float res = params.Restitution;
			res = (res > 0) ? (res*fabs(d)/(dt*length(vel))) : 0.0f;
			vel -= (1+res)*n*dot(n, vel);
			pos = cp;
		}
	}
#endif

	// ���͂̋��E�Ƃ̏Փ˔���
	float3 l0 = params.BoundaryMin;
	float3 l1 = params.BoundaryMax;
	collisionPointAABB(pos, 0.5*(l1+l0), 0.5*(l1-l0), cp, d, n);

	if(d < 0.0){
		float res = params.Restitution;
		res = (res > 0) ? (res*fabs(d)/(dt*length(vel))) : 0.0f;
		vel -= (1+res)*n*dot(n, vel);
		pos = cp;
	}
}

__device__
inline bool calCollisionPolygon(float3 &pos0, float3 &pos1, float3 &vel, float3 v0, float3 v1, float3 v2, float dt)
{
	float3 cp, n;
	if(intersectSegmentTriangle(pos0, pos1, v0, v1, v2, cp, n, params.ParticleRadius) == 1){
		float d = length(pos1-cp);
		n = normalize(n);

		float res = params.Restitution;
		res = (res > 0) ? (res*fabs(d)/(dt*length(vel))) : 0.0f;
		float3 vr = -(1+res)*n*dot(n, vel);

		float l = length(pos1-pos0);
		pos1 = cp+vr*(dt*d/l);
		vel += vr;

		return true;
	}
	return false;
}



/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V(Leap-Frog)
 * @param[inout] ppos �p�[�e�B�N���ʒu
 * @param[inout] pvel �p�[�e�B�N�����x
 * @param[in] pfrc �p�[�e�B�N���ɂ������
 * @param[in] dens �p�[�e�B�N�����x
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
__global__
void sphIntegrate(float4* ppos,	float4* pvel, 
				  float4* pfrc, float* dens, float dt, uint nprts)
{
	uint index = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= nprts) return;

	float3 x = make_float3(ppos[index]);
	float3 v = make_float3(pvel[index]);
	float3 f = make_float3(pfrc[index]);
	//float3 v_old = v;

	float dens0 = dens[index];

	// �X�V�ʒu�C���x�̍X�V
	v += dt*f/dens0;
	x += dt*v;

	// �ő́E���E�Ƃ̏Փ�
	calCollisionSolid(x, v, dt);

	// �ʒu�Ƒ��x�̍X�V
	ppos[index] = make_float4(x);
	pvel[index] = make_float4(v);
}



/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V(Leap-Frog)
 * @param[inout] ppos �p�[�e�B�N���ʒu
 * @param[inout] pvel �p�[�e�B�N�����x
 * @param[in] pfrc �p�[�e�B�N���ɂ������
 * @param[in] dens �p�[�e�B�N�����x
 * @param[in] vrts
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
__global__
void sphIntegrateWithPolygon(float4* ppos, float4* pvel, float4* pfrc, float* dens, 
							 float3* vrts, int3* tris, int tri_num, float dt, rxParticleCell cell)
{
	uint index = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;

	float3 x = make_float3(ppos[index]);
	float3 v = make_float3(pvel[index]);
	float3 f = make_float3(pfrc[index]);
	//float3 v_old = v;
	float3 x_old = x;

	float dens0 = dens[index];

	// �X�V�ʒu�C���x�̍X�V
	v += dt*f/dens0;
	x += dt*v;

	// �|���S���I�u�W�F�N�g�Ƃ̏Փ�
	int3 gridPos[2];
	gridPos[0] = calcGridPos(x_old);	// �ʒu�X�V�O�̃p�[�e�B�N����������O���b�h
	gridPos[1] = calcGridPos(x);		// �ʒu�X�V��̃p�[�e�B�N����������O���b�h
	for(int i = 0; i < 2; ++i){
		uint grid_hash = calcGridHash(gridPos[i]);
		uint start_index = cell.dPolyCellStart[grid_hash];
		if(start_index != 0xffffffff){	// �Z������łȂ����̃`�F�b�N

			uint end_index = cell.dPolyCellEnd[grid_hash];
			for(uint j = start_index; j < end_index; ++j){
				uint pidx = cell.dSortedPolyIdx[j];

				int3 idx = tris[pidx];
				if(calCollisionPolygon(x_old, x, v, vrts[idx.x], vrts[idx.y], vrts[idx.z], dt)){
				}
			}
		}
	}

	// �ő́E���E�Ƃ̏Փ�
	calCollisionSolid(x, v, dt);

	// �ʒu�Ƒ��x�̍X�V
	ppos[index] = make_float4(x);
	pvel[index] = make_float4(v);
}




/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋������疧�x���v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] pos �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float calDensityCellG(int3 gridPos, float3 pos0, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float h = params.EffectiveRadius;
	float d = 0.0f;
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);

		for(uint j = startIndex; j < endIndex; ++j){
			//if(j != index){
				float3 pos1 = make_float3(FETCHC(dSortedPos, j));

				float3 rij = pos0-pos1;
				float r = length(rij);

				if(r <= h){
					float q = h*h-r*r;

					d += params.Mass*params.Wpoly6*q*q*q;
				}

			//}
		}
	}

	return d;
}

/*!
 * �O���b�h��ł̖��x���v�Z
 * @param[out] GridD �O���b�h���x
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] gnum �O���b�h��
 * @param[in] gmin �O���b�h�ŏ����W
 * @param[in] glen �O���b�h��
 */
__global__
void sphCalDensityInGrid(float* GridD, rxParticleCell cell, 
					uint3 gnum, float3 gmin, float3 glen)
{
	uint blockId = __mul24(blockIdx.y, gridDim.x)+blockIdx.x;
	uint i = __mul24(blockId, blockDim.x)+threadIdx.x;

	uint3 gridPos = calcGridPosU(i, gnum);

	if(gridPos.x < gnum.x && gridPos.y < gnum.y && gridPos.z < gnum.z){
		float3 gpos;
		gpos.x = gmin.x+(gridPos.x)*glen.x;
		gpos.y = gmin.y+(gridPos.y)*glen.y;
		gpos.z = gmin.z+(gridPos.z)*glen.z;

		float d = 0.0f;

		int3 pgpos = calcGridPos(gpos);

		float h = params.EffectiveRadius;
		int3 grid_pos0, grid_pos1;
		grid_pos0 = calcGridPos(gpos-make_float3(h));
		grid_pos1 = calcGridPos(gpos+make_float3(h));

		for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
			for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
				for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
					int3 neighbourPos = make_int3(x, y, z);

					d += calDensityCellG(neighbourPos, gpos, cell);
				}
			}
		}

		GridD[gridPos.x+gridPos.y*gnum.x+gridPos.z*gnum.x*gnum.y] = d;
	}

}



//-----------------------------------------------------------------------------
// MARK:Anisotropic Kernel
//-----------------------------------------------------------------------------
/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋�������d�ݕt�����ς��v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] i �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos0 �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] h �T�����a
 * @return �Z�����̃p�[�e�B�N������v�Z�����d�ݕt�����ϒl
 */
__device__
float4 calWeighedAvgPositionCell(int3 gridPos, uint i, float3 pos0, float h, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float4 posw = make_float4(0.0f);	
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);
		for(uint j = startIndex; j < endIndex; ++j){
			//if(j == i) continue;

			float3 pos1 = make_float3(FETCHC(dSortedPos, j));

			float3 rij = pos0-pos1;
			float r = length(rij);

			if(r < h){
				float q = (1.0f-r/h);
				float wij = q*q*q;
				posw += make_float4(pos1*wij, wij);
			}
		}
	}

	return posw;
}

/*!
 * �J�[�l�����S�ʒu�̕������Əd�ݕt�����ς̌v�Z(�J�[�l���֐�)
 * @param[out] upPos �������J�[�l�����S
 * @param[out] wPos �d�ݕt�����σp�[�e�B�N�����W 
 * @param[in]  lambda �������̂��߂̒萔
 * @param[in]  h �T�����a
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void sphCalUpdatedPosition(float4* upPos, float4* wPos, float lambda, float h, rxParticleCell cell)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	float3 pos = make_float3(FETCHC(dSortedPos, index));	// �p�[�e�B�N���ʒu

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos-make_float3(h));
	grid_pos1 = calcGridPos(pos+make_float3(h));

	// ���͂̃O���b�h���܂߂ċߖT�T���C�d�ݕt�����ψʒu���v�Z
	float4 posw  = make_float4(0.0f);
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				posw += calWeighedAvgPositionCell(n_grid_pos, index, pos, h, cell);
			}
		}
	}

	// ���ʂ���������
	uint oIdx = cell.dSortedIndex[index];
	posw.x /= posw.w;
	posw.y /= posw.w;
	posw.z /= posw.w;
	posw.w = 0.0f;

	wPos[oIdx] = posw;
	upPos[oIdx]  = make_float4((1.0-lambda)*pos+lambda*make_float3(posw));
}


/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋�������Covariance Matrix���v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] i �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos0 �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] h �T�����a
 * @return �Z�����̃p�[�e�B�N������v�Z�����d��
 */
__device__
float2 calCovarianceMatrixCell(int3 gridPos, uint i, float3 pos0, float h, matrix3x3 &c, float4 xiw, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float2 wn = make_float2(0.0f);
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);
		for(uint j = startIndex; j < endIndex; ++j){
			//if(j == i) continue;

			float3 pos1 = make_float3(FETCHC(dSortedPos, j));

			float3 rij = pos0-pos1;
			float r = length(rij);

			if(r < h){
				float q = (1.0f-r/h);
				float wij = q*q*q;

				float3 dxj = pos1-make_float3(xiw);

				c.e[0].x += wij*dxj.x*dxj.x;
				c.e[0].y += wij*dxj.x*dxj.y;
				c.e[0].z += wij*dxj.x*dxj.z;
				c.e[1].x += wij*dxj.y*dxj.x;
				c.e[1].y += wij*dxj.y*dxj.y;
				c.e[1].z += wij*dxj.y*dxj.z;
				c.e[2].x += wij*dxj.z*dxj.x;
				c.e[2].y += wij*dxj.z*dxj.y;
				c.e[2].z += wij*dxj.z*dxj.z;
				wn.x += wij;
				wn.y += 1.0;
			}
		}
	}

	return wn;
}


/*!
 * Covariance Matrix�̌v�Z
 * @param[out] PosW �d�ݕt�����σp�[�e�B�N�����W 
 * @param[out] CMat Covariance Matrix
 * @param[in]  lambda �������̂��߂̒萔
 * @param[in]  h �T�����a
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void sphCalCovarianceMatrix(float4* PosW, matrix3x3 *CMat, float h, rxParticleCell cell)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	float3 pos = make_float3(FETCHC(dSortedPos, index));	// �p�[�e�B�N���ʒu

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos-make_float3(h));
	grid_pos1 = calcGridPos(pos+make_float3(h));

	// ���͂̃O���b�h���܂߂ċߖT�T���C�d�ݕt�����ψʒu���v�Z
	float4 xiw  = make_float4(0.0f);
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				xiw += calWeighedAvgPositionCell(n_grid_pos, index, pos, h, cell);
			}
		}
	}

	xiw.x /= xiw.w;
	xiw.y /= xiw.w;
	xiw.z /= xiw.w;

	//__syncthreads();

	// �s��̗v�f�̏�����
	matrix3x3 c;
	for(int k = 0; k < 3; ++k){
		c.e[k].x = 0.0;
		c.e[k].y = 0.0;
		c.e[k].z = 0.0;
	}

	// ���͂̃O���b�h���܂߂ċߖT�T���CCovariance Matrix���v�Z
	float2 wn = make_float2(0.0f);
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				wn += calCovarianceMatrixCell(n_grid_pos, index, pos, h, c, xiw, cell);
			}
		}
	}

	for(int k = 0; k < 3; ++k){
		c.e[k].x /= wn.x;
		c.e[k].y /= wn.x;
		c.e[k].z /= wn.x;
	}

	xiw.w = wn.y;

	// ���ʂ���������
	uint oIdx = cell.dSortedIndex[index];
	PosW[oIdx] = xiw;
	CMat[oIdx] = c;
}

__device__
inline float RxPythag(const float a, const float b)
{
	float absa = abs(a), absb = abs(b);
	return (absa > absb ? absa*(float)sqrt((double)(1.0+(absb/absa)*(absb/absa))) :
		   (absb == 0.0 ? 0.0 : absb*(float)sqrt((double)(1.0+(absa/absb)*(absa/absb)))));
}

//! �ŏ��l����(2�l)
__device__
inline float RXD_MIN(const float &a, const float &b){ return ((a < b) ? a : b); }

//! �ő�l����(2�l)
__device__
inline float RXD_MAX(const float &a, const float &b){ return ((a > b) ? a : b); }

//! a�̕�����b�̕����ɂ��킹��
__device__
inline float RXD_SIGN2(const float &a, const float &b){ return b >= 0 ? (a >= 0 ? a : -a) : (a >= 0 ? -a : a); }

__device__
int svdecomp3(float w[3], float u[9], float v[9], float eps)
{
	bool flag;
	int i, its, j, jj, k, l, nm;
	float anorm, c, f, g, h, s, scale, x, y, z;
	float rv1[3];
	g = scale = anorm = 0.0;
	for(i = 0; i < 3; ++i){
		l = i+2;
		rv1[i] = scale*g;
		g = s = scale = 0.0;
		for(k = i; k < 3; ++k) scale += abs(u[k*3+i]);
		if(scale != 0.0){
			for(k = i; k < 3; ++k){
				u[k*3+i] /= scale;
				s += u[k*3+i]*u[k*3+i];
			}
			f = u[i*3+i];
			g = -RXD_SIGN2(sqrt(s), f);
			h = f*g-s;
			u[i*3+i] = f-g;
			for(j = l-1; j < 3; ++j){
				for(s = 0.0, k = i; k < 3; ++k) s += u[k*3+i]*u[k*3+j];
				f = s/h;
				for(k = i; k < 3; ++k) u[k*3+j] += f*u[k*3+i];
			}
			for(k = i; k < 3; ++k) u[k*3+i] *= scale;
		}

		w[i] = scale*g;
		g = s = scale = 0.0;
		if(i+1 <= 3 && i+1 != 3){
			for(k = l-1; k < 3; ++k) scale += abs(u[i*3+k]);
			if(scale != 0.0){
				for(k = l-1; k < 3; ++k){
					u[i*3+k] /= scale;
					s += u[i*3+k]*u[i*3+k];
				}
				f = u[i*3+l-1];
				g = -RXD_SIGN2(sqrt(s), f);
				h = f*g-s;
				u[i*3+l-1] = f-g;
				for(k = l-1; k < 3; ++k) rv1[k] = u[i*3+k]/h;
				for(j = l-1; j < 3; ++j){
					for(s = 0.0,k = l-1; k < 3; ++k) s += u[j*3+k]*u[i*3+k];
					for(k = l-1; k < 3; ++k) u[j*3+k] += s*rv1[k];
				}
				for(k = l-1; k < 3; ++k) u[i*3+k] *= scale;
			}
		}
		anorm = RXD_MAX(anorm, (abs(w[i])+abs(rv1[i])));
	}
	for(i = 2; i >= 0; --i){
		if(i < 2){
			if(g != 0.0){
				for(j = l; j < 3; ++j){
					v[j*3+i] = (u[i*3+j]/u[i*3+l])/g;
				}
				for(j = l; j < 3; ++j){
					for(s = 0.0, k = l; k < 3; ++k) s += u[i*3+k]*v[k*3+j];
					for(k = l; k < 3; ++k) v[k*3+j] += s*v[k*3+i];
				}
			}
			for(j = l; j < 3; ++j) v[i*3+j] = v[j*3+i] = 0.0;
		}
		v[i*3+i] = 1.0;
		g = rv1[i];
		l = i;
	}
	for(i = 2; i >= 0; --i){
		l = i+1;
		g = w[i];
		for(j = l; j < 3; ++j) u[i*3+j] = 0.0;
		if(g != 0.0){
			g = 1.0/g;
			for(j = l; j < 3; ++j){
				for(s = 0.0, k = l; k < 3; ++k) s += u[k*3+i]*u[k*3+j];
				f = (s/u[i*3+i])*g;
				for(k = i; k < 3; ++k) u[k*3+j] += f*u[k*3+i];
			}
			for(j = i; j < 3; ++j) u[j*3+i] *= g;
		}
		else{
			for(j = i; j < 3; ++j) u[j*3+i] = 0.0;
		}
		++u[i*3+i];
	}
	for(k = 2; k >= 0; --k){
		for(its = 0; its < 30; ++its){
			flag = true;
			for(l = k; l >= 0; --l){
				nm = l-1;
				if(l == 0 || abs(rv1[l]) <= eps*anorm){
					flag = false;
					break;
				}
				if(abs(w[nm]) <= eps*anorm) break;
			}
			if(flag){
				c = 0.0;
				s = 1.0;
				for(i = l; i < k+1; ++i){
					f = s*rv1[i];
					rv1[i] = c*rv1[i];
					if(abs(f) <= eps*anorm) break;
					g = w[i];
					h = RxPythag(f, g);
					w[i] = h;
					h = 1.0/h;
					c = g*h;
					s = -f*h;
					for(j = 0; j < 3; ++j){
						y = u[j*3+nm];
						z = u[j*3+i];
						u[j*3+nm] = y*c+z*s;
						u[j*3+i] = z*c-y*s;
					}
				}
			}
			z = w[k];
			if(l == k){
				if(z < 0.0){
					w[k] = -z;
					for(j = 0; j < 3; ++j) v[j*3+k] = -v[j*3+k];
				}
				break;
			}
			if(its == 29){
				//printf("no convergence in 30 svdcmp iterations");
				return 0;
			}
			x = w[l];
			nm = k-1;
			y = w[nm];
			g = rv1[nm];
			h = rv1[k];
			f = ((y-z)*(y+z)+(g-h)*(g+h))/(2.0*h*y);
			g = RxPythag(f, 1.0f);
			f = ((x-z)*(x+z)+h*((y/(f+RXD_SIGN2(g, f)))-h))/x;
			c = s = 1.0;
			for(j = l; j <= nm; ++j){
				i = j+1;
				g = rv1[i];
				y = w[i];
				h = s*g;
				g = c*g;
				z = RxPythag(f, h);
				rv1[j] = z;
				c = f/z;
				s = h/z;
				f = x*c+g*s;
				g = g*c-x*s;
				h = y*s;
				y *= c;
				for(jj = 0; jj < 3; ++jj){
					x = v[jj*3+j];
					z = v[jj*3+i];
					v[jj*3+j] = x*c+z*s;
					v[jj*3+i] = z*c-x*s;
				}
				z = RxPythag(f, h);
				w[j] = z;
				if(z){
					z = 1.0/z;
					c = f*z;
					s = h*z;
				}
				f = c*g+s*y;
				x = c*y-s*g;
				for(jj = 0; jj < 3; ++jj){
					y = u[jj*3+j];
					z = u[jj*3+i];
					u[jj*3+j] = y*c+z*s;
					u[jj*3+i] = z*c-y*s;
				}
			}
			rv1[l] = 0.0;
			rv1[k] = f;
			w[k] = x;
		}
	}

	// reorder
	int inc = 1;
	float sw;
	float su[3], sv[3];

	do{
		inc *= 3;
		inc++; 
	}while(inc <= 3);

	do{
		inc /= 3;
		for(i = inc; i < 3; ++i){
			sw = w[i];
			for(k = 0; k < 3; ++k) su[k] = u[k*3+i];
			for(k = 0; k < 3; ++k) sv[k] = v[k*3+i];
			j = i;
			while (w[j-inc] < sw){
				w[j] = w[j-inc];
				for(k = 0; k < 3; ++k) u[k*3+j] = u[k*3+j-inc];
				for(k = 0; k < 3; ++k) v[k*3+j] = v[k*3+j-inc];
				j -= inc;
				if (j < inc) break;
			}
			w[j] = sw;
			for(k = 0; k < 3; ++k) u[k*3+j] = su[k];
			for(k = 0; k < 3; ++k) v[k*3+j] = sv[k];

		}
	}while(inc > 1);

	for(k = 0; k < 3; ++k){
		s = 0;
		for(i = 0; i < 3; ++i) if(u[i*3+k] < 0.) s++;
		for(j = 0; j < 3; ++j) if(v[j*3+k] < 0.) s++;
		if(s > 3){
			for(i = 0; i < 3; ++i) u[i*3+k] = -u[i*3+k];
			for(j = 0; j < 3; ++j) v[j*3+k] = -v[j*3+k];
		}
	}

	return 1;
}

/*!
 * ���ْl�����ɂ��ŗL�l���v�Z
 * @param[in]  CMat Covariance Matrix
 * @param[in]  posw �d�ݕt�����ψʒu
 * @param[out] eigen �ŗL�l
 * @param[out] RMat �ŗL�x�N�g��(��]�s��)
 * @param[in]  numParticles �p�[�e�B�N����
 */
__global__
void sphSVDecomposition(matrix3x3 *CMat, float4* posw, float3* eigen, matrix3x3 *RMat, uint numParticles)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= numParticles) return;

	int n = (int)posw[index].w;
	matrix3x3 tmp = CMat[index];
	float u[9], v[9], w[3];
	for(int i = 0; i < 3; ++i){
		u[i*3+0] = tmp.e[i].x;
		u[i*3+1] = tmp.e[i].y;
		u[i*3+2] = tmp.e[i].z;
	}

	// ���ْl����
	svdecomp3(w, u, v, 1.0e-10);
	
	float3 sigma;
	sigma.x = w[0];
	sigma.y = w[1];
	sigma.z = w[2];
	for(int i = 0; i < 3; ++i){
		tmp.e[i].x = u[i*3+0];
		tmp.e[i].y = u[i*3+1];
		tmp.e[i].z = u[i*3+2];
	}
	
	int ne = 10;
	float ks = 1400;
	float kn = 0.5;
	float kr = 4.0;
	if(n > ne){
		float s0 = sigma.x/kr;
		sigma.y = (sigma.y >= s0 ? sigma.y : s0);
		sigma.z = (sigma.z >= s0 ? sigma.z : s0);
		sigma *= ks;
	}
	else{
		sigma = make_float3(kn*1.0f);
	}

	eigen[index] = sigma;
	RMat[index]  = tmp;
}

__device__
inline float calDeterminant(matrix3x3 &mat)
{
	float d = mat.e[0].x*mat.e[1].y*mat.e[2].z- 
			  mat.e[0].x*mat.e[2].y*mat.e[1].z+ 
			  mat.e[1].x*mat.e[2].y*mat.e[0].z- 
			  mat.e[1].x*mat.e[0].y*mat.e[2].z+ 
			  mat.e[2].x*mat.e[0].y*mat.e[1].z- 
			  mat.e[2].x*mat.e[1].y*mat.e[0].z;
	return d;
}

__device__
inline matrix3x3 calInverse(matrix3x3 &mat)
{
	matrix3x3 inv_mat;

	float d = mat.e[0].x*mat.e[1].y*mat.e[2].z- 
			  mat.e[0].x*mat.e[2].y*mat.e[1].z+ 
			  mat.e[1].x*mat.e[2].y*mat.e[0].z- 
			  mat.e[1].x*mat.e[0].y*mat.e[2].z+ 
			  mat.e[2].x*mat.e[0].y*mat.e[1].z- 
			  mat.e[2].x*mat.e[1].y*mat.e[0].z;

	if(d == 0) d = 1;

	inv_mat.e[0].x =  (mat.e[1].y*mat.e[2].z-mat.e[1].z*mat.e[2].y)/d;
	inv_mat.e[1].x = -(mat.e[0].y*mat.e[2].z-mat.e[0].z*mat.e[2].y)/d;
	inv_mat.e[2].x =  (mat.e[0].y*mat.e[1].z-mat.e[0].z*mat.e[1].y)/d;
	inv_mat.e[0].y = -(mat.e[1].x*mat.e[2].z-mat.e[1].z*mat.e[2].x)/d;
	inv_mat.e[1].y =  (mat.e[0].x*mat.e[2].z-mat.e[0].z*mat.e[2].x)/d;
	inv_mat.e[2].y = -(mat.e[0].x*mat.e[1].z-mat.e[0].z*mat.e[1].x)/d;
	inv_mat.e[0].z =  (mat.e[1].x*mat.e[2].y-mat.e[1].y*mat.e[2].x)/d;
	inv_mat.e[1].z = -(mat.e[0].x*mat.e[2].y-mat.e[0].y*mat.e[2].x)/d;
	inv_mat.e[2].z =  (mat.e[0].x*mat.e[1].y-mat.e[0].y*mat.e[1].x)/d;

	return inv_mat;
}

/*!
 * �ŗL�l�C�ŗL�x�N�g��(��]�s��)����ό`�s����v�Z
 * @param[in]  eigen �ŗL�l
 * @param[in]  RMat �ŗL�x�N�g��(��]�s��)
 * @param[out] GMat �ό`�s��
 * @param[in]  numParticles �p�[�e�B�N����
 */
__global__
void sphCalTransformMatrix(float3* eigen, matrix3x3 *RMat, matrix3x3 *GMat, uint numParticles)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= numParticles) return;

	float3 sigma = eigen[index];
	matrix3x3 R = RMat[index];
	matrix3x3 G;

	for(int j = 0; j < 3; ++j){
		G.e[j].x = R.e[j].x*R.e[0].x/sigma.x+R.e[j].y*R.e[0].y/sigma.y+R.e[j].z*R.e[0].z/sigma.z;
		G.e[j].y = R.e[j].x*R.e[1].x/sigma.x+R.e[j].y*R.e[1].y/sigma.y+R.e[j].z*R.e[1].z/sigma.z;
		G.e[j].z = R.e[j].x*R.e[2].x/sigma.x+R.e[j].y*R.e[2].y/sigma.y+R.e[j].z*R.e[2].z/sigma.z;
	}

	float max_diag = -100000000.0f;
	if(G.e[0].x > max_diag) max_diag = G.e[0].x;
	if(G.e[1].y > max_diag) max_diag = G.e[1].y;
	if(G.e[2].z > max_diag) max_diag = G.e[2].z;

	for(int j = 0; j < 3; ++j){
		G.e[j].x /= max_diag;
		G.e[j].y /= max_diag;
		G.e[j].z /= max_diag;
	}

	//G = calInverse(G);

	GMat[index] = G;
}



/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋������疧�x���v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] pos �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float calDensityAnisoCellG(int3 gridPos, float3 pos0, float h, matrix3x3 *GMat, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float d = 0.0f;
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);

		for(uint j = startIndex; j < endIndex; ++j){
			//if(j != index){
				float3 pos1 = make_float3(FETCHC(dSortedPos, j));
				uint jdx = cell.dSortedIndex[j];

				float3 rij = pos0-pos1;
				matrix3x3 Gj = GMat[jdx];
				float3 rg;
				rg.x = Gj.e[0].x*rij.x+Gj.e[0].y*rij.y+Gj.e[0].z*rij.z;
				rg.y = Gj.e[1].x*rij.x+Gj.e[1].y*rij.y+Gj.e[1].z*rij.z;
				rg.z = Gj.e[2].x*rij.x+Gj.e[2].y*rij.y+Gj.e[2].z*rij.z;

				float r = length(rg);

				if(r <= h){
					float q = h*h-r*r;
					float detG = calDeterminant(Gj);

					d += detG*params.Mass*params.Wpoly6*q*q*q;
				}

			//}
		}
	}

	return d;
}

/*!
 * �O���b�h��ł̖��x���v�Z
 * @param[out] GridD �O���b�h���x
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] gnum �O���b�h��
 * @param[in] gmin �O���b�h�ŏ����W
 * @param[in] glen �O���b�h��
 */
__global__
void sphCalDensityAnisoInGrid(float* GridD, matrix3x3 *GMat, float Emax, 
							  rxParticleCell cell, uint3 gnum, float3 gmin, float3 glen)
{
	uint blockId = __mul24(blockIdx.y, gridDim.x)+blockIdx.x;
	uint i = __mul24(blockId, blockDim.x)+threadIdx.x;

	uint3 gridPos = calcGridPosU(i, gnum);

	if(gridPos.x < gnum.x && gridPos.y < gnum.y && gridPos.z < gnum.z){
		float3 gpos;
		gpos.x = gmin.x+(gridPos.x)*glen.x;
		gpos.y = gmin.y+(gridPos.y)*glen.y;
		gpos.z = gmin.z+(gridPos.z)*glen.z;

		matrix3x3 G = GMat[i];
		float detG = calDeterminant(G);
		int3 pgpos = calcGridPos(gpos);

		float h = params.EffectiveRadius;
		int3 grid_pos0, grid_pos1;
		grid_pos0 = calcGridPos(gpos-make_float3(h*3));
		grid_pos1 = calcGridPos(gpos+make_float3(h*3));

		float P = 0.0f;
		for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
			for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
				for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
					int3 neighbourPos = make_int3(x, y, z);

					P += calDensityAnisoCellG(neighbourPos, gpos, h, GMat, cell);
				}
			}
		}

		GridD[gridPos.x+gridPos.y*gnum.x+gridPos.z*gnum.x*gnum.y] = P;
	}

}


#endif // #ifndef _RX_CUSPH_KERNEL_CU_



