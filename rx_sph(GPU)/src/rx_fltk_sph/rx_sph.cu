#include "hip/hip_runtime.h"
/*! 
  @file rx_sph.cu
	
  @brief CUDA�ɂ��SPH

*/
// FILE --rx_sph.cu--


//-----------------------------------------------------------------------------
// �C���N���[�h�t�@�C��
//-----------------------------------------------------------------------------
#include <cstdio>
#include <GL/glew.h>

#include <GL/freeglut.h>

#include "rx_sph_kernel.cu"

//#include "rx_cu_funcs.cuh"
#include <thrust/device_vector.h>
#include <thrust/scan.h>



//-----------------------------------------------------------------------------
// MARK:�O���[�o���ϐ�
//-----------------------------------------------------------------------------
hipArray *g_caNoiseTile = 0;
float *g_dNoiseTile[3] = {0, 0, 0};
uint g_udNoiseTileSize = 0;
uint g_uNoiseTileNum[3*3] = {0, 0, 0,  0, 0, 0,  0, 0, 0};


//-----------------------------------------------------------------------------
// CUDA�֐�
//-----------------------------------------------------------------------------
extern "C"
{
void CuSetParameters(rxSimParams *hostParams)
{
	// copy parameters to constant memory
	RX_CUCHECK( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(rxSimParams)) );
}

void CuClearData(void)
{
}


// �O���b�h���u���b�N���C�u���b�N���X���b�h���̌v�Z
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
	numThreads = min(blockSize, n);
	numBlocks = DivCeil(n, numThreads);
}


//-----------------------------------------------------------------------------
// MARK:3D SPH
//-----------------------------------------------------------------------------
/*!
 * �����Z���̃n�b�V�����v�Z
 * @param[in] 
 * @return 
 */
void CuCalcHash(uint* dGridParticleHash, uint* dSortedIndex, float* dPos, int nprts)
{
	uint numThreads, numBlocks;
	computeGridSize(nprts, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	calcHashD<<< numBlocks, numThreads >>>(dGridParticleHash,
										   dSortedIndex,
										   (float4*)dPos,
										   nprts);
	
	RX_CUERROR("Kernel execution failed");	// �J�[�l���G���[�`�F�b�N
}

/*!
 * �p�[�e�B�N���z����\�[�g���ꂽ���Ԃɕ��ёւ��C
 * �e�Z���̎n�܂�ƏI���̃C���f�b�N�X������
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] oldPos �p�[�e�B�N���ʒu
 * @param[in] oldVel �p�[�e�B�N�����x
 */
void CuReorderDataAndFindCellStart(rxParticleCell cell, float* oldPos, float* oldVel)
{
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	RX_CUCHECK(hipMemset(cell.dCellStart, 0xffffffff, cell.uNumCells*sizeof(uint)));

#if USE_TEX
	RX_CUCHECK(hipBindTexture(0, dSortedPosTex, oldPos, cell.uNumParticles*sizeof(float4)));
	RX_CUCHECK(hipBindTexture(0, dSortedVelTex, oldVel, cell.uNumParticles*sizeof(float4)));
#endif

	uint smemSize = sizeof(uint)*(numThreads+1);

	// �J�[�l�����s
	reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(cell, (float4*)oldPos, (float4*)oldVel);

	RX_CUERROR("Kernel execution failed: CuReorderDataAndFindCellStartD");
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�

#if USE_TEX
	RX_CUCHECK(hipUnbindTexture(dSortedPosTex));
	RX_CUCHECK(hipUnbindTexture(dSortedVelTex));
#endif
}


/*!
 * �p�[�e�B�N�����x�̌v�Z(�J�[�l���Ăяo��)
 * @param[out] dDens �p�[�e�B�N�����x
 * @param[out] dPres �p�[�e�B�N������
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
void CuSphDensity(float* dDens, float* dPres, rxParticleCell cell)
{
	// MRK:CuSphDensity2D
#if USE_TEX
	RX_CUCHECK(hipBindTexture(0, dSortedPosTex, cell.dSortedPos, cell.uNumParticles*sizeof(float4)));
	RX_CUCHECK(hipBindTexture(0, dCellStartTex, cell.dCellStart, cell.uNumCells*sizeof(uint)));
	RX_CUCHECK(hipBindTexture(0, dCellEndTex, cell.dCellEnd, cell.uNumCells*sizeof(uint)));	
#endif
	//RX_CUCHECK(hipMemset((void*)dNewDens, 0, sizeof(float2)*cell.uNumParticles));

	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	sphCalDensity<<< numBlocks, numThreads >>>(dDens, dPres, cell);

	RX_CUERROR("sphCalDensity kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�

#if USE_TEX
	RX_CUCHECK(hipUnbindTexture(dSortedPosTex));
	RX_CUCHECK(hipUnbindTexture(dCellStartTex));
	RX_CUCHECK(hipUnbindTexture(dCellEndTex));
#endif
}

/*!
 * �p�[�e�B�N���@���̌v�Z
 * @param[out] dNewDens �p�[�e�B�N�����x
 * @param[out] dNewPres �p�[�e�B�N������
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
void CuSphNormal(float* dNrms, float* dDens, rxParticleCell cell)
{
	// MRK:CuSphNormal

	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	sphCalNormal<<< numBlocks, numThreads >>>((float4*)dNrms, dDens, cell);

	RX_CUERROR("sphCalNormal kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�

}

/*!
 * �p�[�e�B�N���ɂ�����͂̌v�Z(�J�[�l���Ăяo��)
 * @param[in] dDens �p�[�e�B�N�����x
 * @param[in] dPres �p�[�e�B�N������
 * @param[out] dFrc �p�[�e�B�N���ɂ������
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] dt ���ԃX�e�b�v��
 */
void CuSphForces(float* dDens, float* dPres, float* dFrc, rxParticleCell cell, float dt)
{
#if USE_TEX
	RX_CUCHECK(hipBindTexture(0, dSortedPosTex, cell.dSortedPos, cell.uNumParticles*sizeof(float4)));
	RX_CUCHECK(hipBindTexture(0, dSortedVelTex, cell.dSortedVel, cell.uNumParticles*sizeof(float4)));
	RX_CUCHECK(hipBindTexture(0, dCellStartTex, cell.dCellStart, cell.uNumCells*sizeof(uint)));
	RX_CUCHECK(hipBindTexture(0, dCellEndTex, cell.dCellEnd, cell.uNumCells*sizeof(uint)));	
#endif

	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	sphCalForces<<< numBlocks, numThreads >>>(dDens, dPres, (float4*)dFrc, cell);

	RX_CUERROR("calForcesSPH kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�

#if USE_TEX
	RX_CUCHECK(hipUnbindTexture(dSortedPosTex));
	RX_CUCHECK(hipUnbindTexture(dSortedVelTex));
	RX_CUCHECK(hipUnbindTexture(dCellStartTex));
	RX_CUCHECK(hipUnbindTexture(dCellEndTex));
#endif
}

/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V
 * @param[inout] pos �p�[�e�B�N���ʒu
 * @param[inout] vel �p�[�e�B�N�����x
 * @param[inout] velOld �O�X�e�b�v�̃p�[�e�B�N�����x
 * @param[in] frc �p�[�e�B�N���ɂ������
 * @param[in] dens �p�[�e�B�N�����x
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
void CuSphIntegrate(float* pos, float* vel, float* frc, float* dens, 
					float dt, uint nprts)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(nprts, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	sphIntegrate<<< numBlocks, numThreads >>>((float4*)pos, (float4*)vel, (float4*)frc, dens, 
											  dt, nprts);
	
	RX_CUERROR("sphIntegrate kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}


/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V
 * @param[inout] pos �p�[�e�B�N���ʒu
 * @param[inout] vel �p�[�e�B�N�����x
 * @param[inout] velOld �O�X�e�b�v�̃p�[�e�B�N�����x
 * @param[in] frc �p�[�e�B�N���ɂ������
 * @param[in] dens �p�[�e�B�N�����x
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
void CuSphIntegrateWithPolygon(float* pos, float* vel, float* frc, float* dens, 
							   float* vrts, int* tris, int tri_num, float dt, rxParticleCell cell)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	sphIntegrateWithPolygon<<< numBlocks, numThreads >>>((float4*)pos, (float4*)vel, (float4*)frc, dens, 
											   (float3*)vrts, (int3*)tris, tri_num, dt, cell);
	
	RX_CUERROR("sphIntegrate kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}

/*!
 * �O���b�h��̖��x���Z�o
 * @param[out] dGridD �O���b�h��̖��x�l
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] nx,ny �O���b�h��
 * @param[in] x0,y0 �O���b�h�ŏ����W
 * @param[in] dx,dy �O���b�h��
 */
void CuSphGridDensity(float *dGridD, rxParticleCell cell, 
					  int nx, int ny, int nz, float x0, float y0, float z0, float dx, float dy, float dz)
{
#if USE_TEX
	RX_CUCHECK(hipBindTexture(0, dSortedPosTex, cell.dSortedPos, cell.uNumParticles*sizeof(float4)));
	RX_CUCHECK(hipBindTexture(0, dCellStartTex, cell.dCellStart, cell.uNumCells*sizeof(uint)));
	RX_CUCHECK(hipBindTexture(0, dCellEndTex, cell.dCellEnd, cell.uNumCells*sizeof(uint)));	
#endif

	uint3  gnum = make_uint3(nx, ny, nz);
	float3 gmin = make_float3(x0, y0, z0);
	float3 glen = make_float3(dx, dy, dz);

	int numcell = gnum.x*gnum.y*gnum.z;

	int threads = 128;
	dim3 grid((numcell+threads-1)/threads, 1, 1);
	if(grid.x > 65535){
		grid.y = (grid.x+32768-1)/32768;
		grid.x = 32768;
	}

	// �J�[�l�����s
	sphCalDensityInGrid<<<grid, threads>>>(dGridD, cell, gnum, gmin, glen);

	RX_CUERROR("Kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�

#if USE_TEX
	RX_CUCHECK(hipUnbindTexture(dSortedPosTex));
	RX_CUCHECK(hipUnbindTexture(dCellStartTex));
	RX_CUCHECK(hipUnbindTexture(dCellEndTex));
#endif
}


//-----------------------------------------------------------------------------
// MARK:Anisotropic Kernel
//-----------------------------------------------------------------------------
/*!
 * �J�[�l�����S�ʒu�̍X�V�Əd�ݕt�����ς̌v�Z(�J�[�l���֐�)
 * @param[out] dUpPos �X�V�J�[�l�����S
 * @param[out] dPosW �d�ݕt�����σp�[�e�B�N�����W 
 * @param[in]  lambda �������̂��߂̒萔
 * @param[in]  h �T�����a
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
void CuSphCalUpdatedPosition(float* dUpPos, float* dPosW, float lambda, float h, rxParticleCell cell)
{
	// MRK:CuSphCalUpdatedPosition
#if USE_TEX
	RX_CUCHECK(hipBindTexture(0, dSortedPosTex, cell.dSortedPos, cell.uNumParticles*sizeof(float4)));
	RX_CUCHECK(hipBindTexture(0, dCellStartTex, cell.dCellStart, cell.uNumCells*sizeof(uint)));
	RX_CUCHECK(hipBindTexture(0, dCellEndTex, cell.dCellEnd, cell.uNumCells*sizeof(uint)));	
#endif

	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	sphCalUpdatedPosition<<< numBlocks, numThreads >>>((float4*)dUpPos, (float4*)dPosW, lambda, h, cell);

	RX_CUERROR("sphCalUpdatedPosition kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�

#if USE_TEX
	RX_CUCHECK(hipUnbindTexture(dSortedPosTex));
	RX_CUCHECK(hipUnbindTexture(dCellStartTex));
	RX_CUCHECK(hipUnbindTexture(dCellEndTex));
#endif
}

/*!
 * �������ʒu�ł̏d�ݕt�����ψʒu�̍Čv�Z��covariance matrix�̌v�Z
 * @param[out] dPosW �d�ݕt�����σp�[�e�B�N�����W 
 * @param[out] dCMat Covariance Matrix
 * @param[in]  h �T�����a
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
void CuSphCalCovarianceMatrix(float* dPosW, float* dCMat, float h, rxParticleCell cell)
{
	// MRK:CuSphCalCovarianceMatrix
#if USE_TEX
	RX_CUCHECK(hipBindTexture(0, dSortedPosTex, cell.dSortedPos, cell.uNumParticles*sizeof(float4)));
	RX_CUCHECK(hipBindTexture(0, dCellStartTex, cell.dCellStart, cell.uNumCells*sizeof(uint)));
	RX_CUCHECK(hipBindTexture(0, dCellEndTex, cell.dCellEnd, cell.uNumCells*sizeof(uint)));	
#endif

	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	sphCalCovarianceMatrix<<< numBlocks, numThreads >>>((float4*)dPosW, (matrix3x3*)dCMat, h, cell);

	RX_CUERROR("sphCalCovarianceMatrix kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�

#if USE_TEX
	RX_CUCHECK(hipUnbindTexture(dSortedPosTex));
	RX_CUCHECK(hipUnbindTexture(dCellStartTex));
	RX_CUCHECK(hipUnbindTexture(dCellEndTex));
#endif
}

/*!
 * ���ْl�����ɂ��ŗL�l���v�Z
 * @param[in]  dC Covariance Matrix
 * @param[in]  dPosW �d�ݕt�����ψʒu
 * @param[out] dEigen �ŗL�l
 * @param[out] dR �ŗL�x�N�g��(��]�s��)
 * @param[in]  numParticles �p�[�e�B�N����
 */
void CuSphSVDecomposition(float* dC, float* dPosW, float* dEigen, float* dR, uint numParticles)
{
	// MRK:CuSphCalTransformMatrix

	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(numParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	sphSVDecomposition<<< numBlocks, numThreads >>>((matrix3x3*)dC, (float4*)dPosW, (float3*)dEigen, (matrix3x3*)dR, numParticles);

	RX_CUERROR("sphCalTransformMatrix kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}

/*!
 * �ŗL�l�C�ŗL�x�N�g��(��]�s��)����ό`�s����v�Z
 * @param[in]  dEigen �ŗL�l
 * @param[in]  dR �ŗL�x�N�g��(��]�s��)
 * @param[out] dG �ό`�s��
 * @param[in]  numParticles �p�[�e�B�N����
 */
void CuSphCalTransformMatrix(float* dEigen, float* dR, float *dG, uint numParticles)
{
	// MRK:CuSphCalTransformMatrix

	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(numParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	sphCalTransformMatrix<<< numBlocks, numThreads >>>((float3*)dEigen, (matrix3x3*)dR, (matrix3x3*)dG, numParticles);

	RX_CUERROR("sphCalTransformMatrix kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}


/*!
 * �O���b�h��̖��x���Z�o
 * @param[out] dGridD �O���b�h��̖��x�l
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] nx,ny �O���b�h��
 * @param[in] x0,y0 �O���b�h�ŏ����W
 * @param[in] dx,dy �O���b�h��
 */
void CuSphGridDensityAniso(float *dGridD, float *dG, float Emax, rxParticleCell cell, 
						   int nx, int ny, int nz, float x0, float y0, float z0, float dx, float dy, float dz)
{
#if USE_TEX
	RX_CUCHECK(hipBindTexture(0, dSortedPosTex, cell.dSortedPos, cell.uNumParticles*sizeof(float4)));
	RX_CUCHECK(hipBindTexture(0, dCellStartTex, cell.dCellStart, cell.uNumCells*sizeof(uint)));
	RX_CUCHECK(hipBindTexture(0, dCellEndTex, cell.dCellEnd, cell.uNumCells*sizeof(uint)));	
#endif

	uint3  gnum = make_uint3(nx, ny, nz);
	float3 gmin = make_float3(x0, y0, z0);
	float3 glen = make_float3(dx, dy, dz);

	int numcell = gnum.x*gnum.y*gnum.z;

	int threads = THREAD_NUM;
	dim3 grid((numcell+threads-1)/threads, 1, 1);
	if(grid.x > 65535){
		grid.y = (grid.x+32768-1)/32768;
		grid.x = 32768;
	}

	// �J�[�l�����s
	sphCalDensityAnisoInGrid<<<grid, threads>>>(dGridD, (matrix3x3*)dG, Emax, cell, gnum, gmin, glen);

	RX_CUERROR("Kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�

#if USE_TEX
	RX_CUCHECK(hipUnbindTexture(dSortedPosTex));
	RX_CUCHECK(hipUnbindTexture(dCellStartTex));
	RX_CUCHECK(hipUnbindTexture(dCellEndTex));
#endif
}


}   // extern "C"
