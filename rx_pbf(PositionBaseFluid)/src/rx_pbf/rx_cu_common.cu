#include "hip/hip_runtime.h"
/*! 
  @file rx_cu_common.cu
	
  @brief CUDA���ʃf�o�C�X�֐�
 
  @author Makoto Fujisawa
  @date 2009-08, 2011-06
*/
// FILE --rx_cu_common.cu--

#ifndef _RX_CU_COMMON_CU_
#define _RX_CU_COMMON_CU_


//-----------------------------------------------------------------------------
// �C���N���[�h�t�@�C��
//-----------------------------------------------------------------------------
#include <stdio.h>
#include <math.h>

#include "hip/hip_vector_types.h"
#include <hip/hip_math_constants.h>

#include "rx_cuda_utils.h"

#include "rx_cu_common.cuh"


// �V�~�����[�V�����p�����[�^(�R���X�^���g������)
__constant__ rxSimParams params;


// AABB�̖@������(6*3)
__constant__ float RXG_AABB_NORMALS[18];


//-----------------------------------------------------------------------------
// �֐�
//-----------------------------------------------------------------------------
__device__ __host__
inline uint calUintPow(uint x, uint y)
{
	uint x_y = 1;
	for(uint i=0; i < y;i++) x_y *= x;
	return x_y;
}

/*!
 * a/b�̌v�Z���ʂ�؂�グ
 * @param[in] a,b a/b
 * @return �؂�グ�����Z����
 */
__device__ __host__
inline uint DivCeil(uint a, uint b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}


/*!
 * [a,b]�ɃN�����v
 * @param[in] x �N�����v���������l
 * @param[in] a,b �N�����v���E
 * @return �N�����v���ꂽ���l
 */
__device__
inline float CuClamp(float x, float a, float b)
{
	return max(a, min(b, x));
}
__device__
inline int CuClamp(int x, int a, int b)
{
	return max(a, min(b, x));
}

/*!
 * �[������ for float3
 * @param[in] v �l
 */
__device__
inline int CuIsZero(float3 v)
{
	if(fabsf(v.x) < 1.0e-10 && fabsf(v.y) < 1.0e-10 && fabsf(v.z) < 1.0e-10){
		return 1;
	}
	else{
		return 0;
	}
}

/*!
 * �s��ƃx�N�g���̐�
 * @param[in] m 3x3�s��
 * @param[in] v 3D�x�N�g��
 * @return �ς̌���
 */
__device__
inline float3 CuMulMV(matrix3x3 m, float3 v)
{
	return make_float3(dot(m.e[0], v), dot(m.e[1], v), dot(m.e[2], v));
}



// �O���b�h���u���b�N���C�u���b�N���X���b�h���̌v�Z
__device__ __host__
inline void computeGridSize(uint n, uint thread_per_block, uint &numBlocks, uint &numThreads)
{
	numThreads = min(thread_per_block, n);
	numBlocks = DivCeil(n, numThreads);
}


//-----------------------------------------------------------------------------
// �O���b�h
//-----------------------------------------------------------------------------
/*!
 * �O���b�h�ʒu�v�Z
 * @param[in] p ���W
 * @return �O���b�h���W
 */
__device__ 
inline int3 calcGridPos(float3 p)
{
	int3 gridPos;
	gridPos.x = floor((p.x-params.WorldOrigin.x)/params.CellWidth.x);
	gridPos.y = floor((p.y-params.WorldOrigin.y)/params.CellWidth.y);
	gridPos.z = floor((p.z-params.WorldOrigin.z)/params.CellWidth.z);

	gridPos.x = min(max(gridPos.x, 0), params.GridSize.x-1);
	gridPos.y = min(max(gridPos.y, 0), params.GridSize.y-1);
	gridPos.z = min(max(gridPos.z, 0), params.GridSize.z-1);

	return gridPos;
}

/*!
 * �O���b�h���W����1�����z�񒆂ł̈ʒu���v�Z
 * @param[in] gridPos �O���b�h���W
 * @return �A�h���X
 */
__device__ 
inline uint calcGridHash(int3 gridPos)
{
	return __umul24(__umul24(gridPos.z, params.GridSize.y), params.GridSize.x)+__umul24(gridPos.y, params.GridSize.x)+gridPos.x;
}

/*!
 * �O���b�h�ʒu�v�Z
 * @param[in] p ���W
 * @param[in] origin �O���b�h�̍ŏ����W
 * @param[in] cell_width 1�O���b�h�Z���̕�
 * @param[in] grid_size �O���b�h��
 * @return �O���b�h���W
 */
__device__ 
inline int3 calcGridPosB(float3 p, float3 origin, float3 cell_width, uint3 grid_size)
{
	int3 gridPos;
	gridPos.x = floor((p.x-origin.x)/cell_width.x);
	gridPos.y = floor((p.y-origin.y)/cell_width.y);
	gridPos.z = floor((p.z-origin.z)/cell_width.z);

	gridPos.x = min(max(gridPos.x, 0), grid_size.x-1);
	gridPos.y = min(max(gridPos.y, 0), grid_size.y-1);
	gridPos.z = min(max(gridPos.z, 0), grid_size.z-1);

	return gridPos;
}

/*!
 * �O���b�h���W����1�����z�񒆂ł̈ʒu���v�Z
 * @param[in] gridPos �O���b�h���W
 * @return �A�h���X
 */
__device__ 
inline uint calcGridHashB(int3 gridPos, uint3 grid_size)
{
	return __umul24(__umul24(gridPos.z, grid_size.y), grid_size.x)+__umul24(gridPos.y, grid_size.x)+gridPos.x;
}





//-----------------------------------------------------------------------------
// �A�g�~�b�N�֐�
//-----------------------------------------------------------------------------
#ifdef RX_USE_ATOMIC_FUNC

/*!
 * float��atomicAdd
 */
__device__ 
inline void atomicFloatAdd(float *address, float val)
{
	int i_val = __float_as_int(val);
	int tmp0 = 0;
	int tmp1;
 
	while( (tmp1 = atomicCAS((int *)address, tmp0, i_val)) != tmp0)
	{
		tmp0 = tmp1;
		i_val = __float_as_int(val + __int_as_float(tmp1));
	}
}
/*!
 * double��atomicAdd
 */
__device__ 
inline double atomicDoubleAdd(double *address, double val)
{
	unsigned long long int *address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val+__longlong_as_double(assumed)));
	}while(assumed != old);
	return __longlong_as_double(old);
}
/*!
 * float��atomicMin
 */
__device__ 
inline float atomicFloatMin(float *address, float val)
{
	int *address_as_int = (int*)address;
	int old = atomicMin(address_as_int, __float_as_int(val));
	return __int_as_float(old);
}

/*!
 * float��atomicMax
 */
__device__ 
inline float atomicFloatMax(float *address, float val)
{
	int *address_as_int = (int*)address;
	int old = atomicMax(address_as_int, __float_as_int(val));
	return __int_as_float(old);
}

#endif // #ifdef RX_USE_ATOMIC_FUNC


//-----------------------------------------------------------------------------
// �O���b�h
//-----------------------------------------------------------------------------
/*!
 * 1D�C���f�b�N�X����3D�C���f�b�N�X�ւ̕ϊ�(�O���b�h���͔C��)
 * @param[in] i 1D�C���f�b�N�X
 * @param[in] gridSize �O���b�h��
 * @return 3D�C���f�b�N�X
 */
__device__
inline uint3 calcGridPosU(uint i, uint3 ngrid)
{
	uint3 gridPos;
	uint w = i%(ngrid.x*ngrid.y);
	gridPos.x = w%ngrid.x;
	gridPos.y = w/ngrid.x;
	gridPos.z = i/(ngrid.x*ngrid.y);
	return gridPos;
}
/*!
 * 3D�C���f�b�N�X����1D�C���f�b�N�X�ւ̕ϊ�(�O���b�h���͔C��)
 * @param[in] p 3D�C���f�b�N�X
 * @param[in] gridSize �O���b�h��
 * @return 1D�C���f�b�N�X
 */
__device__
inline uint calcGridPos3(uint3 p, uint3 ngrid)
{
	p.x = min(p.x, ngrid.x-1);
	p.y = min(p.y, ngrid.y-1);
	p.z = min(p.z, ngrid.z-1);
	return (p.z*ngrid.x*ngrid.y)+(p.y*ngrid.x)+p.x;
}



//-----------------------------------------------------------------------------
// ��������
//-----------------------------------------------------------------------------

/*!
 * �����Ɖ~�̌�������(2D, A��)
 * @param[in] A,B �����̗��[�_���W
 * @param[in] C �~�̒��S
 * @param[in] r �~�̔��a
 * @param[out] P ��_���W
 * @return ��_��
 */
__device__ 
static int CuLineCircleIntersection(float2 A, float2 B, float2 C, float r, float2 P[2], float t[2])
{
	float rr = r*r;
	float2 AC = C-A;
	float2 BC = C-B;

	float2 v = B-A;
	float l = length(v);
	v /= l;

	float td = dot(v, AC);
	float2 D = A+td*v;
	float dd = dot(D-C, D-C);

	if(dd < rr){
		float dt = sqrtf(rr-dd);

		float da = rr-dot(AC, AC);
		float db = rr-dot(BC, BC);

		int inter = 0;
		float t1 = td-dt;
		float t2 = td+dt;
		if(t1 >= 0 && t1 <= l){
			P[inter] = A+t1*v;
			t[inter] = t1;
			inter++;
		}
		if(t2 >= 0 && t2 <= l){
			P[inter] = A+t2*v;
			t[inter] = t2;
			inter++;
		}

		return inter;
	}
	else{
		return 0;
	}
}


/*!
 * AABB�Ƌ��̋���
 * @param[in] spos �����S
 * @param[in] r �����a
 * @param[in] sgn
 * @param[in] box_min,box_max AABB�ŏ��C�ő���W�l
 * @param[out] cp AABB�\�ʂ̍ŋߖT�_
 * @param[out] d ���s��AABB�̋���
 * @param[out] n ��_�ɂ�����P�ʖ@���x�N�g��
 */
__device__
inline int collisionSphereAABB(float3 spos, float r, int sgn, float3 box_min, float3 box_max, float3 &cp, float &d, float3 &n)
{
	float3 dist_min;	// box_min�Ƃ̋���
	float3 dist_max;	// box_max�Ƃ̋���
	float d0 = 0.0f;
	float3 n0 = make_float3(0.0f, 0.0f, 0.0f);
	int bout = 0;
	int count = 0;

	// �e�����Ƃɍŏ��ƍő勫�E�O�ɂȂ��Ă��Ȃ������ׂ�
	if((dist_min.x = (spos.x-r)-box_min.x) < 0.0){ bout |= 0x0001; count++; d0 = dist_min.x; n0 = make_float3( 1.0,  0.0,  0.0);}
	if((dist_min.y = (spos.y-r)-box_min.y) < 0.0){ bout |= 0x0002; count++; d0 = dist_min.y; n0 = make_float3( 0.0,  1.0,  0.0);}
	if((dist_min.z = (spos.z-r)-box_min.z) < 0.0){ bout |= 0x0004; count++; d0 = dist_min.z; n0 = make_float3( 0.0,  0.0,  1.0);}
	if((dist_max.x = box_max.x-(spos.x+r)) < 0.0){ bout |= 0x0008; count++; d0 = dist_max.x; n0 = make_float3(-1.0,  0.0,  0.0);}
	if((dist_max.y = box_max.y-(spos.y+r)) < 0.0){ bout |= 0x0010; count++; d0 = dist_max.y; n0 = make_float3( 0.0, -1.0,  0.0);}
	if((dist_max.z = box_max.z-(spos.z+r)) < 0.0){ bout |= 0x0020; count++; d0 = dist_max.z; n0 = make_float3( 0.0,  0.0, -1.0);}

	// �����̓�(�S���ŋ��E��)
	if(bout == 0){
		float min_d = 1e10;
		if(dist_min.x < min_d){ min_d = dist_min.x; n = make_float3( 1.0,  0.0,  0.0); }
		if(dist_min.y < min_d){ min_d = dist_min.y; n = make_float3( 0.0,  1.0,  0.0); }
		if(dist_min.z < min_d){ min_d = dist_min.z; n = make_float3( 0.0,  0.0,  1.0); }

		if(dist_max.x < min_d){ min_d = dist_max.x; n = make_float3(-1.0,  0.0,  0.0); }
		if(dist_max.y < min_d){ min_d = dist_max.y; n = make_float3( 0.0, -1.0,  0.0); }
		if(dist_max.z < min_d){ min_d = dist_max.z; n = make_float3( 0.0,  0.0, -1.0); }

		d = (float)sgn*min_d;
		n *= (float)sgn;
		cp = spos+n*fabs(d);
		return 1;
	}

	// �����̊O
	// sgn = 1:���C-1:�I�u�W�F�N�g
	if(count == 1){
		// ���ʋߖT
		d = (float)sgn*d0;
		n = (float)sgn*n0;
		cp = spos+n*fabs(d);
	}
	else{
		// �G�b�W/�R�[�i�[�ߖT
		float3 x = make_float3(0.0f, 0.0f, 0.0f);
		if(bout & 0x0001) x.x =  dist_min.x;
		if(bout & 0x0002) x.y =  dist_min.y;
		if(bout & 0x0004) x.z =  dist_min.z;
		if(bout & 0x0008) x.x = -dist_max.x;
		if(bout & 0x0010) x.y = -dist_max.y;
		if(bout & 0x0020) x.z = -dist_max.z;

		d = length(x);
		n = normalize(x);

		d *= -(float)sgn;
		n *= -(float)sgn;

		cp = spos+n*fabs(d);

		float3 disp = make_float3(0.00001);
		//Random(disp, 0, 0.00001);
		disp = disp*n;
		cp += disp;
	}

	return 0;
}


/*!
 * AABB�Ɠ_�̋���
 * @param[in] p �_���W
 * @param[in] box_cen AABB�̒��S
 * @param[in] box_ext AABB�̊e�ӂ̒�����1/2
 * @param[out] cp AABB�\�ʂ̍ŋߖT�_
 * @param[out] d ���s��AABB�̋���
 * @param[out] n ��_�ɂ�����P�ʖ@���x�N�g��
 */
__device__
inline int collisionPointAABB(float3 p, float3 box_cen, float3 box_ext, float3 &cp, float &d, float3 &n)
{
	cp = p-box_cen;

	float3 tmp = fabs(cp)-box_ext;
	float res = ((tmp.x > tmp.y && tmp.x > tmp.z) ? tmp.x : (tmp.y > tmp.z ? tmp.y : tmp.z));

	float sgn = (res > 0.0) ? -1.0 : 1.0;

	int coli = 0;
	n = make_float3(0.0f);

	if(cp.x > box_ext.x){
		cp.x = box_ext.x;
		n.x -= 1.0;
		coli++;
	}
	else if(cp.x < -box_ext.x){
		cp.x = -box_ext.x;
		n.x += 1.0;
		coli++;
	}

	if(cp.y > box_ext.y){
		cp.y = box_ext.y;
		n.y -= 1.0;
		coli++;
	}
	else if(cp.y < -box_ext.y){
		cp.y = -box_ext.y;
		n.y += 1.0;
		coli++;
	}

	if(cp.z > box_ext.z){
		cp.z = box_ext.z;
		n.z -= 1.0;
		coli++;
	}
	else if(cp.z < -box_ext.z){
		cp.z = -box_ext.z;
		n.z += 1.0;
		coli++;
	}

	n = normalize(n);

	//if(coli > 1){
	//	float3 disp;
	//	Random(disp, 0, 0.00001);
	//	disp = disp*n;
	//	cp += disp;
	//}

	cp += box_cen;
	d = sgn*length(cp-p);

	return 0;
}


/*!
 * �_��BOX�̋���
 * @param[in] p �_���W
 * @param[in] box_cen BOX�̒��S
 * @param[in] box_ext BOX�̊e�ӂ̒�����1/2
 * @param[in] box_rot BOX�̕����s��(3x3��]�s��)
 * @param[in] box_inv_rot BOX�̕����s��̋t�s��(3x3)
 * @param[out] cp BOX�\�ʂ̍ŋߖT�_
 * @param[out] d �_��BOX�̋���
 * @param[out] n ��_�ɂ�����P�ʖ@���x�N�g��
 */
__device__
inline int collisionPointBox(float3 p, float3 box_cen, float3 box_ext, matrix3x3 box_rot, matrix3x3 box_inv_rot, float3 &cp, float &d, float3 &n)
{
	cp = p-box_cen;
	cp = CuMulMV(box_rot, cp);

	float3 tmp = fabs(cp)-box_ext;

	int coli = 0;
	n = make_float3(0.0f);

	if(tmp.x < 0.0 && tmp.y < 0.0 && tmp.z < 0.0){
		tmp = fabs(tmp);

		if(tmp.x <= tmp.y && tmp.x <= tmp.z){	// x���ʂɋ߂�
			if(cp.x > 0){
				cp.x = box_ext.x;
				n.x += 1.0;
			}
			else{
				cp.x = -box_ext.x;
				n.x -= 1.0;
			}
		}
		else if(tmp.y <= tmp.x && tmp.y <= tmp.z){ // y���ʂɋ߂�
			if(cp.y > 0){
				cp.y = box_ext.y;
				n.y += 1.0;
			}
			else{
				cp.y = -box_ext.y;
				n.y -= 1.0;
			}
		}
		else{ // z���ʂɋ߂�
			if(cp.z > 0){
				cp.z = box_ext.z;
				n.z += 1.0;
			}
			else{
				cp.z = -box_ext.z;
				n.z -= 1.0;
			}
		}

		coli++;
	}

	cp = CuMulMV(box_inv_rot, cp);
	n  = CuMulMV(box_inv_rot, n);

	n = normalize(n);
	cp += box_cen;

	float sgn = (coli) ? -1.0 : 1.0;
	d = sgn*(length(cp-p));

	return 0;
}

/*!
 * �_�Ƌ��̋���
 * @param[in] p �_���W
 * @param[in] sphere_cen ���̒��S
 * @param[in] sphere_rad ���̔��a
 * @param[out] cp �_�Ƌ����S�����Ԑ����Ƌ��̌�_
 * @param[out] d �_�Ƌ��\�ʂ̋���
 * @param[out] n �����S����_�ւ̒P�ʃx�N�g��
 */
__device__
inline int collisionPointSphere(float3 p, float3 sphere_cen, float sphere_rad, float3 &cp, float &d, float3 &n)
{
	n = make_float3(0.0f);

	float3 l = p-sphere_cen;
	float ll = length(l);

	d = ll-sphere_rad;
	if(d < 0.0){
		n = normalize(p-sphere_cen);
		cp = sphere_cen+n*sphere_rad;
	}

	return 0;
}

/*!
 * �_�ƕ��ʂ̋���
 * @param[in] v  �_�̍��W
 * @param[in] px ���ʏ�̓_
 * @param[in] pn ���ʂ̖@��
 * @return ����
 */
__device__ 
inline float distPointPlane(float3 v, float3 px, float3 pn)
{
	return dot((v-px), pn)/length(pn);
}

/*!
 * �O�p�`�Ɠ_�̋����ƍŋߖT�_
 * @param[in] v0,v1,v2	�O�p�`�̒��_
 * @param[in] n			�O�p�`�̖@��
 * @param[in] p			�_
 * @return 
 */
__device__ 
inline int distPointTriangle(float3 v0, float3 v1, float3 v2, float3 n, float3 p, float &dist, float3 &p0)
{
	// �|���S�����܂ޕ��ʂƓ_�̋���
	float l = distPointPlane(p, v0, n);
	
	// ���ʂƂ̍ŋߖT�_���W
	float3 np = p-l*n;

	// �ߖT�_���O�p�`�����ǂ����̔���
	float3 n1 = cross((v0-p), (v1-p));
	float3 n2 = cross((v1-p), (v2-p));
	float3 n3 = cross((v2-p), (v0-p));

	if(dot(n1, n2) > 0 && dot(n2, n3) > 0){
		// �O�p�`��
		dist = l;
		p0 = np;
		return 1;
	}
	else{
		// �O�p�`�O
		return 0;
	}
}


/*!
 * ���C/�����ƎO�p�`�̌���
 * @param[in] P0,P1 ���C/�����̒[�_or���C��̓_
 * @param[in] V0,V1,V2 �O�p�`�̒��_���W
 * @param[out] I ��_���W
 * @retval 1 ��_I�Ō��� 
 * @retval 0 ��_�Ȃ�
 * @retval 2 �O�p�`�̕��ʓ�
 * @retval -1 �O�p�`��"degenerate"�ł���(�ʐς�0�C�܂�C�������_�ɂȂ��Ă���)
 */
inline __device__ 
int intersectSegmentTriangle(float3 P0, float3 P1, 
							 float3 V0, float3 V1, float3 V2, 
							 float3 &I, float3 &n, float rp = 0.01)
{
	// �O�p�`�̃G�b�W�x�N�g���Ɩ@��
	float3 u = V1-V0;		
	float3 v = V2-V0;			
	n = normalize(cross(u, v));
	if(CuIsZero(n)){
		return -1;	// �O�p�`��"degenerate"�ł���(�ʐς�0)
	}

	// ����
	float3 dir = P1-P0;
	float a = dot(n, P0-V0);
	float b = dot(n, dir);
	if(fabs(b) < 1e-10){	// �����ƎO�p�`���ʂ����s
		if(a == 0){
			return 2;	// ���������ʏ�
		}
		else{
			return 0;	// ��_�Ȃ�
		}
	}


	// ��_�v�Z

	// 2�[�_�����ꂼ��قȂ�ʂɂ��邩�ǂ����𔻒�
	float r = -a/b;
	if(a < 0){
		return 0;
	}

	if(r < 0.0){
		return 0;
	}
	else{
		if(fabs(a) > fabs(b)){
			return 0;
		}
		else{
			if(b > 0){
				return 0;
			}
		}
	}

	// �����ƕ��ʂ̌�_
	I = P0+r*dir;

	// ��_���O�p�`���ɂ��邩�ǂ����̔���
	float uu, uv, vv, wu, wv, D;
	uu = dot(u, u);
	uv = dot(u, v);
	vv = dot(v, v);
	float3 w = I-V0;
	wu = dot(w, u);
	wv = dot(w, v);
	D = uv*uv-uu*vv;

	float s, t;
	s = (uv*wv-vv*wu)/D;
	if(s < 0.0 || s > 1.0){
		return 0;
	}
	
	t = (uv*wu-uu*wv)/D;
	if(t < 0.0 || (s+t) > 1.0){
		return 0;
	}

	return 1;
}



/*!
 * AABB�Ɠ_�̋���(�L�����a��)
 * @param[in] spos AABB�̒��S�����_�Ƃ������΍��W�l
 * @param[in] h    �L�����a
 * @param[in] sgn  AABB�̓��ŋ�������:1,�O�Ő�:-1
 * @param[in] vMin AABB�̍ŏ����W�l(���΍��W)
 * @param[in] vMax AABB�̍ő���W�l(���΍��W)
 * @param[out] d   �����t�����l
 * @param[out] n   �ŋߖT�_�̖@������
 * @param[out] p   �L�����a���Ɋ܂�AABB�ʂ̃��X�g(-x,+x,-y,+y,-z,+z��)
 */
__device__ 
inline bool aabb_point_dist(const float3 &spos, const float &r, const int &sgn, 
							const float3 &vMin, const float3 &vMax, 
							float &d, float3 &n)
{
	int bout = 0;
	float d0[6];
	int idx0 = -1;

	// �e�����Ƃɍŏ��ƍő勫�E�O�ɂȂ��Ă��Ȃ������ׂ�
	int c = 0;
	int idx = 0;
	if((d0[idx] = (spos.x-r)-vMin.x) < 0.0){
		bout |= (1 << idx); c++;
		idx0 = idx;
	}
	idx = 1;
	if((d0[idx] = vMax.x-(spos.x+r)) < 0.0){
		bout |= (1 << idx); c++;
		idx0 = idx;
	}
	idx = 2;
	if((d0[idx] = (spos.y-r)-vMin.y) < 0.0){
		bout |= (1 << idx); c++;
		idx0 = idx;
	}
	idx = 3;
	if((d0[idx] = vMax.y-(spos.y+r)) < 0.0){
		bout |= (1 << idx); c++;
		idx0 = idx;
	}
	idx = 4;
	if((d0[idx] = (spos.z-r)-vMin.z) < 0.0){
		bout |= (1 << idx); c++;
		idx0 = idx;
	}
	idx = 5;
	if((d0[idx] = vMax.z-(spos.z+r)) < 0.0){
		bout |= (1 << idx); c++;
		idx0 = idx;
	}

	// AABB��(�S���ŋ��E��)
	if(bout == 0){
		float min_d = 1e10;
		int idx1 = -1;
		for(int i = 0; i < 6; ++i){
			if(d0[i] <= min_d){
				min_d = d0[i];
				idx1 = i;
			}
		}

		d = sgn*min_d;
		n = (idx1 != -1) ? sgn*make_float3(RXG_AABB_NORMALS[3*idx1], RXG_AABB_NORMALS[3*idx1+1], RXG_AABB_NORMALS[3*idx1+2]) : make_float3(0.0);
		return true;
	}

	// AABB�O
	float3 x = make_float3(0.0);
	idx = 0;
	if(bout & (1 << (2*idx))){
		x.x = d0[2*idx];
	}
	else if(bout & (1 << (2*idx+1))){
		x.x = -d0[2*idx+1];
	}
	idx = 1;
	if(bout & (1 << (2*idx))){
		x.y = d0[2*idx];
	}
	else if(bout & (1 << (2*idx+1))){
		x.y = -d0[2*idx+1];
	}
	idx = 2;
	if(bout & (1 << (2*idx))){
		x.z = d0[2*idx];
	}
	else if(bout & (1 << (2*idx+1))){
		x.z = -d0[2*idx+1];
	}

	// sgn = 1:���C-1:�I�u�W�F�N�g
	if(c == 1){
		// ���ʋߖT
		d = sgn*d0[idx0];
		n = sgn*make_float3(RXG_AABB_NORMALS[3*idx0], RXG_AABB_NORMALS[3*idx0+1], RXG_AABB_NORMALS[3*idx0+2]);
	}
	else{
		// �G�b�W/�R�[�i�[�ߖT
		d = -sgn*length(x);
		n = sgn*(-normalize(x));
	}

	return false;
}


/*!
 * AABB�Ɠ_�̋���(�L�����a��)
 * @param[in] spos AABB�̒��S�����_�Ƃ������΍��W�l
 * @param[in] h    �L�����a
 * @param[in] sgn  AABB�̓��ŋ�������:1,�O�Ő�:-1
 * @param[in] vMin AABB�̍ŏ����W�l(���΍��W)
 * @param[in] vMax AABB�̍ő���W�l(���΍��W)
 * @param[out] d   �����t�����l
 * @param[out] n   �ŋߖT�_�̖@������
 * @param[out] p   �L�����a���Ɋ܂�AABB�ʂ̃��X�g(-x,+x,-y,+y,-z,+z��)
 */
__device__ 
inline bool aabb_point_dist(const float3 &spos, const float &h, const int &sgn, 
							const float3 &vMin, const float3 &vMax, 
							float &d, float3 &n, int &np, int &plist, float pdist[6])
{
	int bout = 0;
	float d0[6];
	int idx0 = -1;
	plist = 0;

	// �e�����Ƃɍŏ��ƍő勫�E�O�ɂȂ��Ă��Ȃ������ׂ�
	int c = 0;
	int idx = 0;
	if((d0[idx] = spos.x-vMin.x) < 0.0){
		bout |= (1 << idx); c++;
		idx0 = idx;
	}
	idx = 1;
	if((d0[idx] = vMax.x-spos.x) < 0.0){
		bout |= (1 << idx); c++;
		idx0 = idx;
	}
	idx = 2;
	if((d0[idx] = spos.y-vMin.y) < 0.0){
		bout |= (1 << idx); c++;
		idx0 = idx;
	}
	idx = 3;
	if((d0[idx] = vMax.y-spos.y) < 0.0){
		bout |= (1 << idx); c++;
		idx0 = idx;
	}
	idx = 4;
	if((d0[idx] = spos.z-vMin.z) < 0.0){
		bout |= (1 << idx); c++;
		idx0 = idx;
	}
	idx = 5;
	if((d0[idx] = vMax.z-spos.z) < 0.0){
		bout |= (1 << idx); c++;
		idx0 = idx;
	}

	// AABB��(�S���ŋ��E��)
	if(bout == 0){
		np = 0;
		float min_d = 1e10;
		int idx1 = -1;
		for(int i = 0; i < 6; ++i){
			if(d0[i] <= min_d){
				min_d = d0[i];
				idx1 = i;
			}
			if(d0[i] < h){
				plist |= (1 << i);
				pdist[i] = d0[i];
				np++;
			}
		}

		d = sgn*min_d;
		n = (idx1 != -1) ? sgn*make_float3(RXG_AABB_NORMALS[3*idx1], RXG_AABB_NORMALS[3*idx1+1], RXG_AABB_NORMALS[3*idx1+2]) : make_float3(0.0);
		return true;
	}

	// AABB�O
	np = 0;
	float3 x = make_float3(0.0);
	idx = 0;
	if(bout & (1 << (2*idx))){
		x.x = d0[2*idx];
		if(-d0[2*idx] < h){
			plist |= (1 << (2*idx));
			pdist[2*idx] = d0[2*idx];
			np++;
		}
	}
	else if(bout & (1 << (2*idx+1))){
		x.x = -d0[2*idx+1];
		if(-d0[2*idx+1] < h){
			plist |= (1 << (2*idx+1));
			pdist[2*idx+1] = d0[2*idx+1];
			np++;
		}
	}
	idx = 1;
	if(bout & (1 << (2*idx))){
		x.y = d0[2*idx];
		if(-d0[2*idx] < h){
			plist |= (1 << (2*idx));
			pdist[2*idx] = d0[2*idx];
			np++;
		}
	}
	else if(bout & (1 << (2*idx+1))){
		x.y = -d0[2*idx+1];
		if(-d0[2*idx+1] < h){
			plist |= (1 << (2*idx+1));
			pdist[2*idx+1] = d0[2*idx+1];
			np++;
		}
	}
	idx = 2;
	if(bout & (1 << (2*idx))){
		x.z = d0[2*idx];
		if(-d0[2*idx] < h){
			plist |= (1 << (2*idx));
			pdist[2*idx] = d0[2*idx];
			np++;
		}
	}
	else if(bout & (1 << (2*idx+1))){
		x.z = -d0[2*idx+1];
		if(-d0[2*idx+1] < h){
			plist |= (1 << (2*idx+1));
			pdist[2*idx+1] = d0[2*idx+1];
			np++;
		}
	}

	// sgn = 1:���C-1:�I�u�W�F�N�g
	if(c == 1){
		// ���ʋߖT
		d = sgn*d0[idx0];
		n = sgn*make_float3(RXG_AABB_NORMALS[3*idx0], RXG_AABB_NORMALS[3*idx0+1], RXG_AABB_NORMALS[3*idx0+2]);
	}
	else{
		// �G�b�W/�R�[�i�[�ߖT
		d = -sgn*length(x);
		n = sgn*(-normalize(x));
	}

	return false;
}




/*!
 * �����Ƌ��̌�������
 * @param[in] s0,s1	�����̒[�_
 * @param[in] sc,r   ���̒��S���W�Ɣ��a
 * @param[out] d2 �����Ƃ̋����̓��
 * @return ���������true
 */
__device__ 
inline bool segment_sphere(const float3 &s0, const float3 &s1, const float3 &sc, const float &r, float &d2)
{
	float3 v = s1-s0;
	float3 c = sc-s0;

	float vc = dot(v, c);
	if(vc < 0){		// ���̒��S�������̎n�_s0�̊O�ɂ���
		d2 = dot(c, c);
		return (d2 < r*r);	// �����S�Ǝn�_s0�̋����Ō�������
	}
	else{
		float v2 = dot(v, v);
		if(vc > v2){	// ���̒��S�������̏I�_s1�̊O�ɂ���
			d2 = dot(s1-sc, s1-sc);
			return (d2 < r*r);	// �����S�ƏI�_s1�̋����Ō�������
		}
		else{			// ����s0��s1�̊Ԃɂ���
			float3 a = (vc*v)/dot(v, v)-c;
			d2 = dot(a, a);
			return (d2 < r*r);	// �����Ƌ����S�̋����Ō�������
		}
	}
}

/*!
 * ����(���܂ޒ���)�Ɠ_�̋���
 * @param[in] v0,v1 �����̗��[�_���W
 * @param[in] p �_�̍��W
 * @return ����
 */
__device__ 
inline double segment_point_dist(const float3 &v0, const float3 &v1, const float3 &p)
{
	float3 v = normalize(v1-v0);
	float3 vp = p-v0;
	float3 vh = dot(vp, v)*v;
	return length(vp-vh);
}


/*!
 * ����(���C,������)�Ƌ��̌�������
 * @param[in] p,d ���C�̌��_�ƕ���
 * @param[in] c,r ���̒��S�Ɣ��a
 * @param[out] t1,t2 p�����_�܂ł̋���
 * @return ��_��
 */
__device__
inline int ray_sphere(const float3 &p, const float3 &d, const float3 &sc, const float r, float &t1, float &t2)
{
	float3 q = p-sc;	// �����S���W�n�ł̌������_���W

	float a = dot(d, d);
	float b = 2*dot(q, d);
	float c = dot(q, q)-r*r;

	// ���ʎ�
	float D = b*b-4*a*c;

	if(D < 0.0){ // �����Ȃ�
		return 0;
	}
	else if(D < 1e-8){ // ��_��1
		t1 = -b/(2*a);
		t2 = -1;
		return 1;
	}
	else{ // ��_��2
		float sqrtD = sqrt(D);
		t1 = (-b-sqrtD)/(2*a);
		t2 = (-b+sqrtD)/(2*a);
		return 2;
	}

}
/*!
 * �O�p�`�Ƌ��̌�������
 * @param[in] v0,v1,v2	�O�p�`�̒��_
 * @param[in] n			�O�p�`�̖@��
 * @param[in] p			�ŋߖT�_
 * @return 
 */
__device__
inline bool triangle_sphere(const float3 &v0, const float3 &v1, const float3 &v2, const float3 &n, 
							const float3 &c, const float &r, float &dist, float3 &ipoint)
{
	// �|���S�����܂ޕ��ʂƋ����S�̋���
	float d = dot(v0, n);
	float l = dot(n, c)-d;

	dist = l;
	if(l > r) return false;

	// ���ʂƂ̍ŋߖT�_���W
	float3 p = c-l*n;

	// �ߖT�_���O�p�`�����ǂ����̔���
	float3 n1 = cross((v0-c), (v1-c));
	float3 n2 = cross((v1-c), (v2-c));
	float3 n3 = cross((v2-c), (v0-c));

	ipoint = p;
	dist = l;
	if(dot(n1, n2) > 0 && dot(n2, n3) > 0){		// �O�p�`��
		return true;
	}
	else{		// �O�p�`�O
		// �O�p�`�̊e�G�b�W�Ƌ��̏Փ˔���
		for(int e = 0; e < 3; ++e){
			float3 va0 = (e == 0 ? v0 : (e == 1 ? v1 : v2));
			float3 va1 = (e == 0 ? v1 : (e == 1 ? v2 : v0));

			float t1, t2;
			int n = ray_sphere(va0, normalize(va1-va0), c, r, t1, t2);

			if(n){
				float le2 = dot(va1-va0, va1-va0);
				if((t1 >= 0.0 && t1*t1 < le2) || (t2 >= 0.0 && t2*t2 < le2)){
					return true;
				}
			}
		}
		return false;
	}
}


#endif // #ifndef _RX_CU_COMMON_CU_



