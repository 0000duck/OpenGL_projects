#include "hip/hip_runtime.h"
/*! 
  @file rx_sph.cu
	
  @brief CUDA�ɂ��SPH

  @author Makoto Fujisawa
  @date 2009-08, 2011-06
*/
// FILE --rx_sph.cu--


//-----------------------------------------------------------------------------
// �C���N���[�h�t�@�C��
//-----------------------------------------------------------------------------
#include <cstdio>
#include <GL/glew.h>
#include <GL/glut.h>

#include "rx_pbf_kernel.cu"

#include <thrust/device_vector.h>
#include <thrust/scan.h>



//-----------------------------------------------------------------------------
// CUDA�֐�
//-----------------------------------------------------------------------------
extern "C"
{
	
void CuSetParameters(rxSimParams *hostParams)
{
	// copy parameters to constant memory
	RX_CUCHECK( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(rxSimParams)) );
}

void CuClearData(void)
{
}

/*!
 * thrust::exclusive_scan�̌Ăяo��
 * @param[out] dScanData scan��̃f�[�^
 * @param[in] dData ���f�[�^
 * @param[in] num �f�[�^��
 */
void CuScanf(float* dScanData, float* dData, unsigned int num)
{
	thrust::exclusive_scan(thrust::device_ptr<float>(dData), 
						   thrust::device_ptr<float>(dData+num),
						   thrust::device_ptr<float>(dScanData));
}


/*!
 * �����Z���̃n�b�V�����v�Z
 * @param[in] 
 * @return 
 */
void CuCalcHash(uint* dGridParticleHash, uint* dSortedIndex, float* dPos, int nprts)
{
	uint numThreads, numBlocks;
	computeGridSize(nprts, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	calcHashD<<< numBlocks, numThreads >>>(dGridParticleHash,
										   dSortedIndex,
										   (float4*)dPos,
										   nprts);
	
	RX_CUERROR("calcHashD kernel execution failed");	// �J�[�l���G���[�`�F�b�N
}


/*!
 * �p�[�e�B�N���z����\�[�g���ꂽ���Ԃɕ��ёւ��C
 * �e�Z���̎n�܂�ƏI���̃C���f�b�N�X������
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] oldPos �p�[�e�B�N���ʒu
 * @param[in] oldVel �p�[�e�B�N�����x
 */
void CuReorderDataAndFindCellStart(rxParticleCell cell, float* oldPos, float* oldVel)
{
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	RX_CUCHECK(hipMemset(cell.dCellStart, 0xffffffff, cell.uNumCells*sizeof(uint)));

	uint smemSize = sizeof(uint)*(numThreads+1);

	// �J�[�l�����s
	reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(cell, (float4*)oldPos, (float4*)oldVel);

	RX_CUERROR("reorderDataAndFindCellStartD kernel execution failed");
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}


/*!
 * �����Z���̃n�b�V�����v�Z
 * @param[in] 
 * @return 
 */
void CuCalcHashB(uint* dGridParticleHash, uint* dSortedIndex, float* dPos, 
				 float3 world_origin, float3 cell_width, uint3 grid_size, int nprts)
{
	uint numThreads, numBlocks;
	computeGridSize(nprts, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	calcHashB<<< numBlocks, numThreads >>>(dGridParticleHash,
										   dSortedIndex,
										   (float4*)dPos,
										   world_origin, 
										   cell_width, 
										   grid_size, 
										   nprts);
	
	RX_CUERROR("Kernel execution failed : calcHashB");	// �J�[�l���G���[�`�F�b�N
}

/*!
 * �p�[�e�B�N���z����\�[�g���ꂽ���Ԃɕ��ёւ��C
 * �e�Z���̎n�܂�ƏI���̃C���f�b�N�X������
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] oldPos �p�[�e�B�N���ʒu
 */
void CuReorderDataAndFindCellStartB(rxParticleCell cell, float* oldPos)
{
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	RX_CUCHECK(hipMemset(cell.dCellStart, 0xffffffff, cell.uNumCells*sizeof(uint)));

	uint smemSize = sizeof(uint)*(numThreads+1);

	// �J�[�l�����s
	reorderDataAndFindCellStartB<<< numBlocks, numThreads, smemSize>>>(cell, (float4*)oldPos);

	RX_CUERROR("Kernel execution failed: CuReorderDataAndFindCellStartB");
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}










//-----------------------------------------------------------------------------
// ���E�p�[�e�B�N������
//-----------------------------------------------------------------------------
/*!
 * ���E�p�[�e�B�N���̑̐ς��v�Z
 *  - "Versatile Rigid-Fluid Coupling for Incompressible SPH", 2.2 ��(3)�̏�
 * @param[out] dVolB ���E�p�[�e�B�N���̑̐�
 * @param[in]  mass �p�[�e�B�N������
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
void CuSphBoundaryVolume(float* dVolB, float mass, rxParticleCell cell)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	sphCalBoundaryVolume<<< numBlocks, numThreads >>>(dVolB, cell);

	RX_CUERROR("kernel execution failed : sphCalBoundaryVolume");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}

/*!
 * �p�[�e�B�N�����x�̌v�Z(�J�[�l���Ăяo��)
 * @param[out] dDens �p�[�e�B�N�����x
 * @param[out] dPres �p�[�e�B�N������
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
void CuSphBoundaryDensity(float* dDens, float* dPres, float* dPos, float* dVolB, rxParticleCell bcell, uint pnum)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(pnum, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	sphCalBoundaryDensity<<< numBlocks, numThreads >>>(dDens, dPres, (float4*)dPos, dVolB, bcell, pnum);

	RX_CUERROR("kernel execution failed : sphCalBoundaryDensity");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}

/*!
 * �p�[�e�B�N���ɂ�����͂̌v�Z(�J�[�l���Ăяo��)
 * @param[in] dDens �p�[�e�B�N�����x
 * @param[in] dPres �p�[�e�B�N������
 * @param[out] dFrc �p�[�e�B�N���ɂ������
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] dt ���ԃX�e�b�v��
 */
void CuSphBoundaryForces(float* dDens, float* dPres, float* dPos, float* dVolB, float* dFrc, rxParticleCell bcell, uint pnum)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(pnum, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	sphCalBoundaryForce<<< numBlocks, numThreads >>>(dDens, dPres, (float4*)dPos, dVolB, (float4*)dFrc, bcell, pnum);

	RX_CUERROR("kernel execution failed : sphCalBoundaryForce");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}





//-----------------------------------------------------------------------------
// PBF
//-----------------------------------------------------------------------------

/*!
 * �p�[�e�B�N�����x�̌v�Z(�J�[�l���Ăяo��)
 * @param[out] dDens �p�[�e�B�N�����x
 * @param[out] dPres �p�[�e�B�N������
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
void CuPbfDensity(float* dDens, rxParticleCell cell)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfCalDensity<<< numBlocks, numThreads >>>(dDens, cell);

	RX_CUERROR("pbfCalDensity kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}

/*!
 * �p�[�e�B�N���ɂ�����͂̌v�Z(�J�[�l���Ăяo��)
 * @param[in] dDens �p�[�e�B�N�����x
 * @param[out] dFrc �p�[�e�B�N���ɂ������
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] dt ���ԃX�e�b�v��
 */
void CuPbfExternalForces(float* dDens, float* dFrc, rxParticleCell cell, float dt)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfCalExternalForces<<< numBlocks, numThreads >>>(dDens, (float4*)dFrc, cell);

	RX_CUERROR("pbfCalExternalForces kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}

/*!
 * �X�P�[�����O�t�@�N�^�̌v�Z
 * @param[in] dPos �p�[�e�B�N�����S���W
 * @param[out] dDens �p�[�e�B�N�����x
 * @param[out] dScl �X�P�[�����O�t�@�N�^
 * @param[in] eps �ɘa�W��
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 */
void CuPbfScalingFactor(float* dPos, float* dDens, float* dScl, float eps, rxParticleCell cell)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfCalScalingFactor<<< numBlocks, numThreads >>>((float4*)dPos, dDens, dScl, eps, cell);

	RX_CUERROR("pbfCalScalingFactor kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}

/*!
 * ���ϖ��x�ϓ��̌v�Z
 *  - ���ׂẴp�[�e�B�N�����x�̏������x�Ƃ̍����J�[�l���Ōv�Z���CPrefix Sum (Scan)�ł��̍��v�����߂�
 * @param[out] dErrScan �ϓ��l��Scan���ʂ��i�[����z��
 * @param[out] dErr �p�[�e�B�N�����x�ϓ��l
 * @param[in] dDens �p�[�e�B�N�����x
 * @param[in] rest_dens �������x
 * @param[in] nprts �p�[�e�B�N����
 * @return ���ϖ��x�ϓ�
 */
float CuPbfCalDensityFluctuation(float* dErrScan, float* dErr, float* dDens, float rest_dens, uint nprts)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(nprts, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfDensityFluctuation<<< numBlocks, numThreads >>>(dErr, dDens, rest_dens, nprts);

	RX_CUERROR("pbfDensityFluctuation kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�

	// �e�p�[�e�B�N���̖��x�ϓ���Scan
	CuScanf(dErrScan, dErr, nprts);

	// Exclusive scan (�Ō�̗v�f��0�Ԗڂ���n-2�Ԗڂ܂ł̍��v�ɂȂ��Ă���)�Ȃ̂ŁC
	// Scan�O�z��̍Ō�(n-1�Ԗ�)�̗v�f�ƍ��v���邱�ƂŖ��x�ϓ��̍��v���v�Z
	float lval, lsval;
	RX_CUCHECK(hipMemcpy((void*)&lval, (void*)(dErr+nprts-1), sizeof(float), hipMemcpyDeviceToHost));
	RX_CUCHECK(hipMemcpy((void*)&lsval, (void*)(dErrScan+nprts-1), sizeof(float), hipMemcpyDeviceToHost));
	float dens_var = lval+lsval;

	return dens_var/(float)nprts;
}

/*!
 * �ʒu�C���ʂ̌v�Z
 * @param[in] dPos �p�[�e�B�N�����S���W
 * @param[in] dScl �X�P�[�����O�t�@�N�^
 * @param[out] dDp �ʒu�C����
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 */
void CuPbfPositionCorrection(float* dPos, float* dScl, float* dDp, rxParticleCell cell)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfPositionCorrection<<< numBlocks, numThreads >>>((float4*)dPos, dScl, (float4*)dDp, cell);

	RX_CUERROR("pbfPositionCorrection kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}

/*!
 * �p�[�e�B�N���ʒu���X�V
 * @param[inout] dPos �p�[�e�B�N���ʒu
 * @param[in] dDp �ʒu�C����
 * @param[in] nprts �p�[�e�B�N����
 */
void CuPbfCorrectPosition(float* dPos, float* dDp, uint nprts)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(nprts, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfCorrectPosition<<< numBlocks, numThreads >>>((float4*)dPos, (float4*)dDp, nprts);
	
	RX_CUERROR("pbfCorrectPosition kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}



/*!
 * ���E�p�[�e�B�N�����x���]���̃p�[�e�B�N�����x�ɉ�����
 * @param[inout] dDens ���̃p�[�e�B�N�����x
 * @param[in] dPos  ���̃p�[�e�B�N������
 * @param[in] dVolB ���E�p�[�e�B�N���̐�
 * @param[in] bcell ���E�p�[�e�B�N���O���b�h�f�[�^
 */
void CuPbfBoundaryDensity(float* dDens, float* dPos, float* dVolB, rxParticleCell bcell, uint pnum)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(pnum, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfCalBoundaryDensity<<< numBlocks, numThreads >>>(dDens, (float4*)dPos, dVolB, bcell, pnum);

	RX_CUERROR("pbfCalBoundaryDensity kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}


/*!
 * �X�P�[�����O�t�@�N�^�̌v�Z(���E�p�[�e�B�N���܂�)
 * @param[in] dPos ���̃p�[�e�B�N�����S���W
 * @param[out] dDens ���̃p�[�e�B�N�����x
 * @param[out] dScl ���̃p�[�e�B�N���̃X�P�[�����O�t�@�N�^
 * @param[in] eps �ɘa�W��
 * @param[in] cell ���̃p�[�e�B�N���O���b�h�f�[�^
 * @param[in] dVolB ���E�p�[�e�B�N���̐�
 * @param[out] dSclB ���E�p�[�e�B�N���̃X�P�[�����O�t�@�N�^
 * @param[in] bcell ���E�p�[�e�B�N���O���b�h�f�[�^
 */
void CuPbfScalingFactorWithBoundary(float* dPos, float* dDens, float* dScl, float eps, rxParticleCell cell, 
									   float* dVolB, float* dSclB, rxParticleCell bcell)
{
	// ���̃p�[�e�B�N���̐������X���b�h�𗧂Ă�
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfCalScalingFactorWithBoundary<<< numBlocks, numThreads >>>((float4*)dPos, dDens, dScl, eps, cell, dVolB, bcell);

	RX_CUERROR("pbfCalScalingFactorWithBoundary kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�

	// ���E�p�[�e�B�N���̃X�P�[�����O�t�@�N�^�̌v�Z
	// ���E�p�[�e�B�N���̐������X���b�h�𗧂Ă�
	computeGridSize(bcell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfCalBoundaryScalingFactor<<< numBlocks, numThreads >>>((float4*)dPos, dDens, eps, cell, dVolB, dSclB, bcell);

	RX_CUERROR("pbfCalScalingFactorWithBoundary kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�

}

/*!
 * �ʒu�C���ʂ̌v�Z(���E�p�[�e�B�N���܂�)
 * @param[in] dPos ���̃p�[�e�B�N�����S���W
 * @param[in] dScl ���̃p�[�e�B�N���̃X�P�[�����O�t�@�N�^
 * @param[out] dDens ���̃p�[�e�B�N���ʒu�C����
 * @param[in] cell ���̃p�[�e�B�N���O���b�h�f�[�^
 * @param[in] dVolB ���E�p�[�e�B�N���̐�
 * @param[in] dSclB ���E�p�[�e�B�N���̃X�P�[�����O�t�@�N�^
 * @param[in] bcell ���E�p�[�e�B�N���O���b�h�f�[�^
 */
void CuPbfPositionCorrectionWithBoundary(float* dPos, float* dScl, float* dDp, rxParticleCell cell, 
											float* dVolB, float* dSclB, rxParticleCell bcell)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfPositionCorrectionWithBoundary<<< numBlocks, numThreads >>>((float4*)dPos, dScl, (float4*)dDp, cell, 
																	  dVolB, dSclB, bcell);

	RX_CUERROR("pbfPositionCorrectionWithBoundary kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}



/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V
 *  - �O�p�`�|���S�����E��
 *  - ���݂̈ʒu�̂ݎg���ďՓ˔���
 * @param[in] dPos �p�[�e�B�N���ʒu
 * @param[in] dVel �p�[�e�B�N�����x
 * @param[in] dAcc �p�[�e�B�N�������x
 * @param[out] dNewPos �X�V���ꂽ�p�[�e�B�N���ʒu
 * @param[out] dNewVel �X�V���ꂽ�p�[�e�B�N�����x
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
void CuPbfIntegrate(float* dPos, float* dVel, float* dAcc, 
					float* dNewPos, float* dNewVel, float dt, uint nprts)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(nprts, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfIntegrate<<< numBlocks, numThreads >>>((float4*)dPos, (float4*)dVel, (float4*)dAcc, 
												 (float4*)dNewPos, (float4*)dNewVel, dt, nprts);
	
	RX_CUERROR("pbfIntegrate kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}

/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V
 *  - �O�p�`�|���S�����E��
 *  - ���݂̈ʒu�̂ݎg���ďՓ˔���
 * @param[in] dPos �p�[�e�B�N���ʒu
 * @param[in] dVel �p�[�e�B�N�����x
 * @param[in] dAcc �p�[�e�B�N�������x
 * @param[out] dNewPos �X�V���ꂽ�p�[�e�B�N���ʒu
 * @param[out] dNewVel �X�V���ꂽ�p�[�e�B�N�����x
 * @param[in] dVrts �O�p�`�|���S�����_
 * @param[in] dTris �O�p�`�|���S���C���f�b�N�X
 * @param[in] tri_num �O�p�`�|���S����
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 */
void CuPbfIntegrateWithPolygon(float* dPos, float* dVel, float* dAcc, 
							   float* dNewPos, float* dNewVel, 
							   float* dVrts, int* dTris, int tri_num, float dt, rxParticleCell cell)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfIntegrateWithPolygon<<< numBlocks, numThreads >>>((float4*)dPos, (float4*)dVel, (float4*)dAcc, 
															(float4*)dNewPos, (float4*)dNewVel, (float3*)dVrts, (int3*)dTris, tri_num, dt, cell);
	
	RX_CUERROR("pbfIntegrateWithPolygon kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}


/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V
 *  - ���݂̈ʒu�ƏC����̈ʒu��2���g���ďՓ˔���(PBF�������Ɏg�p)
 * @param[in] dPos �p�[�e�B�N���ʒu
 * @param[in] dVel �p�[�e�B�N�����x
 * @param[in] dAcc �p�[�e�B�N�������x
 * @param[out] dNewPos �X�V���ꂽ�p�[�e�B�N���ʒu
 * @param[out] dNewVel �X�V���ꂽ�p�[�e�B�N�����x
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
void CuPbfIntegrate2(float* dPos, float* dVel, float* dAcc, 
					 float* dNewPos, float* dNewVel, float dt, uint nprts)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(nprts, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfIntegrate2<<< numBlocks, numThreads >>>((float4*)dPos, (float4*)dVel, (float4*)dAcc, 
												  (float4*)dNewPos, (float4*)dNewVel, dt, nprts);
	
	RX_CUERROR("pbfIntegrate2 kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}
/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V
 *  - �O�p�`�|���S�����E��
 *  - ���݂̈ʒu�ƏC����̈ʒu��2���g���ďՓ˔���(PBF�������Ɏg�p)
 * @param[in] dPos �p�[�e�B�N���ʒu
 * @param[in] dVel �p�[�e�B�N�����x
 * @param[in] dAcc �p�[�e�B�N�������x
 * @param[out] dNewPos �X�V���ꂽ�p�[�e�B�N���ʒu
 * @param[out] dNewVel �X�V���ꂽ�p�[�e�B�N�����x
 * @param[in] dVrts �O�p�`�|���S�����_
 * @param[in] dTris �O�p�`�|���S���C���f�b�N�X
 * @param[in] tri_num �O�p�`�|���S����
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 */
void CuPbfIntegrateWithPolygon2(float* dPos, float* dVel, float* dAcc, 
								float* dNewPos, float* dNewVel, 
								float* dVrts, int* dTris, int tri_num, float dt, rxParticleCell cell)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfIntegrateWithPolygon2<<< numBlocks, numThreads >>>((float4*)dPos, (float4*)dVel, (float4*)dAcc, 
															 (float4*)dNewPos, (float4*)dNewVel, (float3*)dVrts, (int3*)dTris, tri_num, dt, cell);
	
	RX_CUERROR("pbfIntegrateWithPolygon2 kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}


/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V
 * @param[in] pos �X�V���ꂽ�p�[�e�B�N���ʒu
 * @param[inout] new_pos �X�e�b�v�ŏ��̃p�[�e�B�N���ʒu/�V�����p�[�e�B�N�����x
 * @param[out] new_vel �V�����p�[�e�B�N�����x
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
void CuPbfUpdatePosition(float* dPos, float* dNewPos, float* dNewVel, float dt, uint nprts)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(nprts, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfUpdatePosition<<< numBlocks, numThreads >>>((float4*)dPos, (float4*)dNewPos, (float4*)dNewVel, dt, nprts);
	
	RX_CUERROR("CupbfUpdatePosition kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}

/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V
 * @param[in] pos �X�V���ꂽ�p�[�e�B�N���ʒu
 * @param[inout] new_pos �X�e�b�v�ŏ��̃p�[�e�B�N���ʒu/�V�����p�[�e�B�N�����x
 * @param[out] new_vel �V�����p�[�e�B�N�����x
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
void CuPbfUpdateVelocity(float* dPos, float* dNewPos, float* dNewVel, float dt, uint nprts)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(nprts, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfUpdateVelocity<<< numBlocks, numThreads >>>((float4*)dPos, (float4*)dNewPos, (float4*)dNewVel, dt, nprts);
	
	RX_CUERROR("pbfUpdateVelocity kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}

/*!
 * XSPH�ɂ��S���v�Z
 * @param[in] dPos �p�[�e�B�N�����S���W
 * @param[in] dVel �p�[�e�B�N�����x
 * @param[out] dNewVel �X�V���ꂽ�p�[�e�B�N�����x
 * @param[in] dDens �p�[�e�B�N�����x
 * @param[in] c �S���v�Z�p�p�����[�^
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 */
void CuXSphViscosity(float* dPos, float* dVel, float* dNewVel, float* dDens, float c, rxParticleCell cell)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	xsphVisocosity<<< numBlocks, numThreads >>>((float4*)dPos, (float4*)dVel, (float4*)dNewVel, dDens, c, cell);

	RX_CUERROR("xsphVisocosity kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}

/*!
 * �O���b�h��̖��x���Z�o
 * @param[out] dGridD �O���b�h��̖��x�l
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] nx,ny,nz �O���b�h��
 * @param[in] x0,y0,z0 �O���b�h�ŏ����W
 * @param[in] dx,dy,dz �O���b�h��
 */
void CuPbfGridDensity(float *dGridD, rxParticleCell cell, 
					  int nx, int ny, int nz, float x0, float y0, float z0, float dx, float dy, float dz)
{
	uint3  gnum = make_uint3(nx, ny, nz);
	float3 gmin = make_float3(x0, y0, z0);
	float3 glen = make_float3(dx, dy, dz);

	int numcell = gnum.x*gnum.y*gnum.z;

	int threads = 128;
	dim3 grid((numcell+threads-1)/threads, 1, 1);
	if(grid.x > 65535){
		grid.y = (grid.x+32768-1)/32768;
		grid.x = 32768;
	}

	// �J�[�l�����s
	pbfCalDensityInGrid<<<grid, threads>>>(dGridD, cell, gnum, gmin, glen);

	RX_CUERROR("pbfCalDensityInGrid Kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�
}

/*!
 * �p�[�e�B�N���@���̌v�Z
 * @param[out] dNrms �p�[�e�B�N���@��
 * @param[int] dDens �p�[�e�B�N�����x
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
void CuPbfNormal(float* dNrms, float* dDens, rxParticleCell cell)
{
	// 1�X���b�h/�p�[�e�B�N��
	uint numThreads, numBlocks;
	computeGridSize(cell.uNumParticles, THREAD_NUM, numBlocks, numThreads);

	// �J�[�l�����s
	pbfCalNormal<<< numBlocks, numThreads >>>((float4*)dNrms, dDens, cell);

	RX_CUERROR("pbfCalNormal kernel execution failed");	// �J�[�l�����s�G���[�`�F�b�N
	RX_CUCHECK(hipDeviceSynchronize());		// �S�ẴX���b�h���I���̂�҂�

}










}   // extern "C"
