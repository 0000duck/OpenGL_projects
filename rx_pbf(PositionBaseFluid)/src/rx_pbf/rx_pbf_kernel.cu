#include "hip/hip_runtime.h"
/*! 
  @file rx_pvsph_kernel.cu
	
  @brief CUDA�ɂ��PVSPH
 
  @author Makoto Fujisawa
  @date 2014-12
*/

#ifndef _RX_PVSPH_KERNEL_CU_
#define _RX_PVSPH_KERNEL_CU_


//-----------------------------------------------------------------------------
// �C���N���[�h�t�@�C��
//-----------------------------------------------------------------------------
#include "rx_cu_common.cu"

//-----------------------------------------------------------------------------
// �n�b�V��
//-----------------------------------------------------------------------------
/*!
 * �e�p�[�e�B�N���̃O���b�h�n�b�V���l�̌v�Z
 * @param[out] gridParticleHash �n�b�V���l
 * @param[out] dSortedIndex �p�[�e�B�N���C���f�b�N�X�l(�\�[�g�O�̏����l���������)
 * @param[in] pos �p�[�e�B�N���ʒu���i�[�����z��
 * @param[in] nprts �p�[�e�B�N����
 */
__global__
void calcHashD(uint*   dGridParticleHash, 
			   uint*   dSortedIndex, 
			   float4* dPos, 
			   uint	   nprts)
{
	uint index = __umul24(blockIdx.x, blockDim.x)+threadIdx.x;
	if(index >= nprts) return;
	
	volatile float4 p = dPos[index];
	int3 gridPos = calcGridPos(make_float3(p.x, p.y, p.z));
	uint hash = calcGridHash(gridPos);

	dGridParticleHash[index] = hash;
	dSortedIndex[index] = index;
}

/*!
 * �e�p�[�e�B�N���̃O���b�h�n�b�V���l
 *  - params���g��Ȃ��ŃO���b�h���������ŗ^����
 * @param[out] gridParticleHash �p�[�e�B�N���̃O���b�h�n�b�V���l
 * @param[out] dSortedIndex �p�[�e�B�N���C���f�b�N�X�l(�\�[�g�O�̏����l���������)
 * @param[in] dPos �p�[�e�B�N���ʒu���i�[�����z��
 * @param[in] world_origin �O���b�h�ŏ����W�l
 * @param[in] cell_width �O���b�h�̃Z����
 * @param[in] grid_size �O���b�h��
 * @param[in] nprts �p�[�e�B�N����
 */
__global__
void calcHashB(uint*   dGridParticleHash, 
			   uint*   dSortedIndex, 
			   float4*  dPos, 
			   float3  world_origin, 
			   float3  cell_width, 
			   uint3   grid_size, 
			   uint	   nprts)
{
	uint index = __umul24(blockIdx.x, blockDim.x)+threadIdx.x;
	if(index >= nprts) return;
	
	float3 p = make_float3(dPos[index]);
	int3 gridPos = calcGridPosB(make_float3(p.x, p.y, p.z), world_origin, cell_width, grid_size);
	uint hash = calcGridHashB(gridPos, grid_size);

	dGridParticleHash[index] = hash;
	dSortedIndex[index] = index;
}


/*!
 * �p�[�e�B�N���f�[�^���\�[�g���āC�n�b�V�����̊e�Z���̍ŏ��̃A�h���X������
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] dSortedPos �p�[�e�B�N���ʒu
 * @param[in] dSortedVel �p�[�e�B�N�����x
 */
__global__
void reorderDataAndFindCellStartD(rxParticleCell cell, float4* dSortedPos, float4* dSortedVel)
{
	extern __shared__ uint sharedHash[];	// �T�C�Y : blockSize+1
	uint index = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
	
	uint hash;
	if(index < cell.uNumParticles){
		hash = cell.dGridParticleHash[index];	// �n�b�V���l

		sharedHash[threadIdx.x+1] = hash;	// �n�b�V���l���V�F�A�[�h�������Ɋi�[

		if(index > 0 && threadIdx.x == 0){
			// �e�V�F�A�[�h�������̍ŏ��ׂ͗̃O���b�h�̃p�[�e�B�N���̃n�b�V���l���i�[
			sharedHash[0] = cell.dGridParticleHash[index-1];
		}
	}

	__syncthreads();
	
	if(index < cell.uNumParticles){
		// �C���f�b�N�X0�ł���C�������́C��O�̃p�[�e�B�N���̃O���b�h�n�b�V���l���قȂ�ꍇ�C
		// �p�[�e�B�N���͕����̈�̍ŏ�
		if(index == 0 || hash != sharedHash[threadIdx.x]){
			cell.dCellStart[hash] = index;
			if(index > 0){
				// ��O�̃p�[�e�B�N���́C��O�̕����̈�̍Ō�
				cell.dCellEnd[sharedHash[threadIdx.x]] = index;
			}
		}

		// �C���f�b�N�X���Ō�Ȃ�΁C�����̈�̍Ō�
		if(index == cell.uNumParticles-1){
			cell.dCellEnd[hash] = index+1;
		}

		// �ʒu�Ƒ��x�̃f�[�^����ёւ�
		// �\�[�g�����C���f�b�N�X�ŎQ�Ƃ��\�����T�����̃O���[�o���������A�N�Z�X���ɗ͗}���邽�߂Ƀf�[�^���̂��̂���ёւ���
		uint sortedIndex = cell.dSortedIndex[index];
		float4 pos = FETCH(dSortedPos, sortedIndex);
		float4 vel = FETCH(dSortedVel, sortedIndex);

		cell.dSortedPos[index] = pos;
		cell.dSortedVel[index] = vel;
	}
}

/*!
 * �p�[�e�B�N���f�[�^���\�[�g���āC�n�b�V�����̊e�Z���̍ŏ��̃A�h���X������
 *  - �ʒu�̂�
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] dPos �p�[�e�B�N���ʒu
 */
__global__
void reorderDataAndFindCellStartB(rxParticleCell cell, float4* dPos)
{
	extern __shared__ uint sharedHash[];	// �T�C�Y : blockSize+1
	uint index = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
	
	uint hash;
	if(index < cell.uNumParticles){
		hash = cell.dGridParticleHash[index];	// �n�b�V���l

		sharedHash[threadIdx.x+1] = hash;	// �n�b�V���l���V�F�A�[�h�������Ɋi�[

		if(index > 0 && threadIdx.x == 0){
			// �e�V�F�A�[�h�������̍ŏ��ׂ͗̃O���b�h�̃p�[�e�B�N���̃n�b�V���l���i�[
			sharedHash[0] = cell.dGridParticleHash[index-1];
		}
	}

	__syncthreads();
	
	if(index < cell.uNumParticles){
		// �C���f�b�N�X0�ł���C�������́C��O�̃p�[�e�B�N���̃O���b�h�n�b�V���l���قȂ�ꍇ�C
		// �p�[�e�B�N���͕����̈�̍ŏ�
		if(index == 0 || hash != sharedHash[threadIdx.x]){
			cell.dCellStart[hash] = index;
			if(index > 0){
				// ��O�̃p�[�e�B�N���́C��O�̕����̈�̍Ō�
				cell.dCellEnd[sharedHash[threadIdx.x]] = index;
			}
		}

		// �C���f�b�N�X���Ō�Ȃ�΁C�����̈�̍Ō�
		if(index == cell.uNumParticles-1){
			cell.dCellEnd[hash] = index+1;
		}

		// �ʒu�Ƒ��x�̃f�[�^����ёւ�
		// �\�[�g�����C���f�b�N�X�ŎQ�Ƃ��\�����T�����̃O���[�o���������A�N�Z�X���ɗ͗}���邽�߂Ƀf�[�^���̂��̂���ёւ���
		uint sortedIndex = cell.dSortedIndex[index];
		float4 pos = dPos[sortedIndex];
		cell.dSortedPos[index] = pos;
	}
}



//-----------------------------------------------------------------------------
// ���E�p�[�e�B�N�������J�[�l��
//-----------------------------------------------------------------------------
/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋������狫�E�p�[�e�B�N���̑̐ς��v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] index �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float calBoundaryVolumeCell(int3 gridPos, uint i, float3 pos0, rxParticleCell cell)
{
	uint gridHash = calcGridHashB(gridPos, params.GridSizeB);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = cell.dCellStart[gridHash];

	float h = params.EffectiveRadius;
	float mw = 0.0f;
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = cell.dCellEnd[gridHash];
		for(uint j = startIndex; j < endIndex; ++j){
			float3 pos1 = make_float3(cell.dSortedPos[j]);

			float3 rij = pos0-pos1;
			float r = length(rij);

			if(r <= h){
				float q = h*h-r*r;
				mw += params.Mass*params.Wpoly6*q*q*q;
			}
		}
	}

	return mw;
}

/*!
 * ���E�p�[�e�B�N���̑̐όv�Z(�J�[�l���֐�)
 * @param[out] newVolB �p�[�e�B�N���̐�
 * @param[in]  cell ���E�p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void sphCalBoundaryVolume(float* newVolB, rxParticleCell cell)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	float3 pos = make_float3(cell.dSortedPos[index]);	// �p�[�e�B�N���ʒu
	//int3 grid_pos = calcGridPos(pos);	// �p�[�e�B�N����������O���b�h�ʒu
	float h = params.EffectiveRadius;

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPosB(pos-make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);
	grid_pos1 = calcGridPosB(pos+make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);


	// ���͂̃O���b�h���܂߂ċߖT�T��
	float mw = 0.0f;
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				mw += calBoundaryVolumeCell(n_grid_pos, index, pos, cell);
			}
		}
	}

	// �̐ς����ʂɏ�������
	uint oIdx = cell.dSortedIndex[index];
	newVolB[oIdx] = params.Mass/mw;
}

/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋������疧�x���v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] i �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos0 �v�Z���W
 * @param[in] dVolB ���E�p�[�e�B�N�����z�̐�
 * @param[in] bcell ���E�p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float calBoundaryDensityCell(int3 gridPos, uint i, float3 pos0, float* dVolB, rxParticleCell bcell)
{
	uint gridHash = calcGridHashB(gridPos, params.GridSizeB);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = bcell.dCellStart[gridHash];

	float h = params.EffectiveRadius;
	float dens = 0.0f;
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = bcell.dCellEnd[gridHash];
		for(uint j = startIndex; j < endIndex; ++j){
			//if(j == i) continue;

			float3 pos1 = make_float3(bcell.dSortedPos[j]);
			uint jdx = bcell.dSortedIndex[j];

			float3 rij = pos0-pos1;
			float r = length(rij);

			if(r <= h){
				float q = h*h-r*r;
				dens += params.Density*dVolB[jdx]*params.Wpoly6*q*q*q;
			}
		}
	}

	return dens;
}

/*!
 * ���E�p�[�e�B�N�����x�v�Z(�J�[�l���֐�)
 * @param[out] newDens ���E�p�[�e�B�N�����x
 * @param[out] newPres ���E�p�[�e�B�N������ - PBF�ł͎g��Ȃ�
 * @param[in] dPos  ���E�p�[�e�B�N���ʒu
 * @param[in] dVolB ���E�p�[�e�B�N�����z�̐�
 * @param[in] bcell ���E�p�[�e�B�N���O���b�h�f�[�^
 * @param[in] pnum  ���E�p�[�e�B�N����
 */
__global__
void sphCalBoundaryDensity(float* newDens, float* newPres, float4* dPos, float* dVolB, rxParticleCell bcell, uint pnum)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= pnum) return;	
	
	float3 pos = make_float3(dPos[index]);	// �p�[�e�B�N���ʒu
	float h = params.EffectiveRadius;

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPosB(pos-make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);
	grid_pos1 = calcGridPosB(pos+make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);

	// ���͂̃O���b�h���܂߂ċߖT�T���C���x�v�Z
	float dens = 0.0f;
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				dens += calBoundaryDensityCell(n_grid_pos, index, pos, dVolB, bcell);
			}
		}
	}

	dens += newDens[index];

	// �K�X�萔���g�������͎Z�o
	float pres;
	pres = params.GasStiffness*(dens-params.Density);

	// ���x�ƈ��͒l�����ʂɏ�������
	newDens[index] = dens;
	newPres[index] = pres;
}


/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋������疧�x���v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] i �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos0 �v�Z���W
 * @param[in] dVolB ���E�p�[�e�B�N�����z�̐�
 * @param[in] dens0 �p�[�e�B�N��i�̖��x
 * @param[in] pres0 �p�[�e�B�N��i�̈���
 * @param[in] bcell ���E�p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�����O��
 */
__device__
float3 calBoundaryForceCell(int3 gridPos, uint i, float3 pos0, float* dVolB, float dens0, float pres0, rxParticleCell bcell)
{
	uint gridHash = calcGridHashB(gridPos, params.GridSizeB);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = bcell.dCellStart[gridHash];

	float h = params.EffectiveRadius;
	float3 bp = make_float3(0.0f);
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = bcell.dCellEnd[gridHash];
		float prsi = pres0/(dens0*dens0);
		for(uint j = startIndex; j < endIndex; ++j){
			float3 pos1 = make_float3(bcell.dSortedPos[j]);
			uint jdx = bcell.dSortedIndex[j];

			float3 rij = pos0-pos1;
			float r = length(rij);

			if(r <= h && r > 0.0001){
				float q = h-r;
				bp += -params.Density*dVolB[jdx]*prsi*params.GWspiky*q*q*rij/r;
			}
		}
	}

	return bp;
}

/*!
 * ���E�p�[�e�B�N���ɂ��͂̌v�Z(�J�[�l���֐�)
 * @param[in] dDens ���E�p�[�e�B�N�����x
 * @param[in] dPres ���E�p�[�e�B�N������
 * @param[in] dPos  ���E�p�[�e�B�N���ʒu
 * @param[in] dVolB ���E�p�[�e�B�N�����z�̐�
 * @param[out] outFrc �O��
 * @param[in] bcell ���E�p�[�e�B�N���O���b�h�f�[�^
 * @param[in] pnum  ���E�p�[�e�B�N����
 */
__global__
void sphCalBoundaryForce(float* dDens, float* dPres, float4* dPos, float* dVolB, float4* outFrc, rxParticleCell bcell, uint pnum)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= pnum) return;	
	
	float3 pos = make_float3(dPos[index]);	// �p�[�e�B�N���ʒu
	float h = params.EffectiveRadius;

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPosB(pos-make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);
	grid_pos1 = calcGridPosB(pos+make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);

	// ���x�ƈ���
	float dens0 = dDens[index];
	float pres0 = dPres[index];

	// ���͂̃O���b�h���܂߂ċߖT�T���C���x�v�Z
	float3 frc = make_float3(0.0f);
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				frc += calBoundaryForceCell(n_grid_pos, index, pos, dVolB, dens0, pres0, bcell);
			}
		}
	}

	// ���x�ƈ��͒l�����ʂɏ�������
	outFrc[index] += make_float4(frc, 0.0f);
}


//-----------------------------------------------------------------------------
// pbf
//-----------------------------------------------------------------------------
/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋������疧�x���v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] i �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos0 �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float calDensityCellPB(int3 gridPos, uint i, float3 pos0, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float h = params.EffectiveRadius;
	float dens = 0.0f;
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);
		for(uint j = startIndex; j < endIndex; ++j){
			//if(j == i) continue;

			float3 pos1 = make_float3(FETCHC(dSortedPos, j));

			float3 rij = pos0-pos1;
			float r = length(rij);

			if(r <= h){
				float q = h*h-r*r;
				dens += params.Mass*params.Wpoly6*q*q*q;
			}
		}
	}

	return dens;
}



/*!
 * �p�[�e�B�N�����x�v�Z(�J�[�l���֐�)
 * @param[out] newDens �p�[�e�B�N�����x
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void pbfCalDensity(float* newDens, rxParticleCell cell)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	float3 pos = make_float3(FETCHC(dSortedPos, index));	// �p�[�e�B�N���ʒu
	//int3 grid_pos = calcGridPos(pos);	// �p�[�e�B�N����������O���b�h�ʒu
	float h = params.EffectiveRadius;

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos-make_float3(h));
	grid_pos1 = calcGridPos(pos+make_float3(h));

	// ���͂̃O���b�h���܂߂ċߖT�T���C���x�v�Z
	float dens = 0.0f;
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				dens += calDensityCellPB(n_grid_pos, index, pos, cell);
			}
		}
	}

	// ���x�ƈ��͒l�����ʂɏ�������
	uint oIdx = cell.dSortedIndex[index];
	newDens[oIdx] = dens;
}

/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋�������͏���v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] i �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos0 �v�Z���W
 * @param[in] vel0 �v�Z���W�̑��x
 * @param[in] dens0 �v�Z���W�̖��x
 * @param[in] dens �p�[�e�B�N�����x
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�����͏�
 */
__device__
float3 calExtForceCell(int3 gridPos, uint i, float3 pos0, float3 vel0, float dens0, float* dens, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float h = params.EffectiveRadius;

	float3 frc = make_float3(0.0f);
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);
		for(uint j = startIndex; j < endIndex; ++j){
			if(j != i){
				// �ߖT�p�[�e�B�N���̃p�����[�^
				float3 pos1 = make_float3(FETCHC(dSortedPos, j));
				float3 vel1 = make_float3(FETCHC(dSortedVel, j));

				float3 rij = pos0-pos1;
				float r = length(rij);

				if(r <= h && r > 0.0001){
					float dens1 = dens[cell.dSortedIndex[j]];

					float3 vij = vel1-vel0;

					float q = h-r;

					// �S����
					frc += params.Viscosity*params.Mass*(vij/dens1)*params.LWvisc*q;
				}
			}
		}
	}

	return frc;
}

/*!
 * �p�[�e�B�N���ɂ�����O�͂̌v�Z(�J�[�l���֐�)
 * @param[in] dens �p�[�e�B�N�����x
 * @param[out] outFrc �p�[�e�B�N���ɂ������
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void pbfCalExternalForces(float* dens, float4* outFrc, rxParticleCell cell)
{
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	// �\�[�g�ςݔz�񂩂�p�[�e�B�N���f�[�^���擾
	float3 pos0 = make_float3(FETCHC(dSortedPos, index));
	float3 vel0 = make_float3(FETCHC(dSortedVel, index));
	float h = params.EffectiveRadius;

	// �p�[�e�B�N���̃\�[�g�Ȃ��z���ł̃C���f�b�N�X
	uint oIdx = cell.dSortedIndex[index];

	float3 frc = make_float3(0.0f);
	float dens0 = dens[oIdx];

	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos0-make_float3(h));
	grid_pos1 = calcGridPos(pos0+make_float3(h));

	// ���͂̃O���b�h���܂߂ċߖT�T���C���͍��C�S�������v�Z
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);

				frc += calExtForceCell(n_grid_pos, index, pos0, vel0, dens0, dens, cell);
			}
		}
	}

	// �O��(�d��)
	frc += params.Gravity;

	outFrc[oIdx] = make_float4(frc, 0.0f);
}


/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋�������X�P�[�����O�t�@�N�^�̕��ꍀ�v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] i �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos0 �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float calScalingFactorCell(int3 gridPos, uint i, float3 pos0, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float h = params.EffectiveRadius;
	float r0 = params.Density;
	float sd = 0.0f;
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);
		for(uint j = startIndex; j < endIndex; ++j){
			if(j == i) continue;

			float3 pos1 = make_float3(FETCHC(dSortedPos, j));

			float3 rij = pos0-pos1;
			float r = length(rij);

			if(r <= h && r > 0.0){
				float q = h-r;

				// Spiky�J�[�l���ňʒu�ϓ����v�Z
				float3 dp = (params.GWspiky*q*q*rij/r)/r0;

				sd += dot(dp, dp);
			}

		}
	}

	return sd;
}

/*!
 * �X�P�[�����O�t�@�N�^�̌v�Z
 * @param[in] ppos �p�[�e�B�N�����S���W
 * @param[out] pdens �p�[�e�B�N�����x
 * @param[out] pscl �X�P�[�����O�t�@�N�^
 * @param[in] eps �ɘa�W��
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void pbfCalScalingFactor(float4* ppos, float* pdens, float* pscl, float eps, rxParticleCell cell)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	float3 pos = make_float3(FETCHC(dSortedPos, index));	// �p�[�e�B�N���ʒu
	//int3 grid_pos = calcGridPos(pos);	// �p�[�e�B�N����������O���b�h�ʒu

	float h = params.EffectiveRadius;
	float r0 = params.Density;

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos-make_float3(h));
	grid_pos1 = calcGridPos(pos+make_float3(h));

	// ���͂̃O���b�h���܂߂ċߖT�T���C���x�v�Z
	float dens = 0.0f;
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				dens += calDensityCellPB(n_grid_pos, index, pos, cell);
			}
		}
	}

	// ���x�S������(��(1))
	float C = dens/r0-1.0;

	// ���͂̃O���b�h���܂߂ċߖT�T���C�X�P�[�����O�t�@�N�^�̕��ꍀ�v�Z
	float sd = 0.0f;
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				sd += calScalingFactorCell(n_grid_pos, index, pos, cell);
			}
		}
	}

	// �p�[�e�B�N���̃\�[�g�Ȃ��z���ł̃C���f�b�N�X
	uint oIdx = cell.dSortedIndex[index];

	// �X�P�[�����O�t�@�N�^�̌v�Z(��(11))
	pscl[oIdx] = -C/(sd+eps);

	// �X�V���ꂽ���x
	pdens[oIdx] = dens;
}


/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋�������X�P�[�����O�t�@�N�^�̕��ꍀ�v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] index �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float3 calPositionCorrectionCell(int3 gridPos, uint i, float3 pos0, float* pscl, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float k = params.AP_K;
	float n = params.AP_N;
	float wq = params.AP_WQ;

	float h = params.EffectiveRadius;
	float r0 = params.Density;
	float3 dp = make_float3(0.0);

	float dt = params.Dt;

	float si = pscl[cell.dSortedIndex[i]];

	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);
		for(uint j = startIndex; j < endIndex; ++j){
			if(j == i) continue;

			float3 pos1 = make_float3(FETCHC(dSortedPos, j));

			float3 rij = pos0-pos1;
			float r = length(rij);

			if(r <= h && r > 0.0){
				float scorr = 0.0f;

				if(params.AP){
					float q1 = h*h-r*r;
					float ww = params.Wpoly6*q1*q1*q1/wq;
					scorr = -k*pow(ww, n)*dt*dt;
				}
				float q = h-r;
				float sj = pscl[cell.dSortedIndex[j]];

				// Spiky�J�[�l���ňʒu�C���ʂ��v�Z
				dp += (si+sj+scorr)*(params.GWspiky*q*q*rij/r)/r0;
			}

		}
	}

	return dp;
}

/*!
 * �X�P�[�����O�t�@�N�^�̌v�Z
 * @param[in] ppos �p�[�e�B�N�����S���W
 * @param[out] pdens �p�[�e�B�N�����x
 * @param[out] pscl �X�P�[�����O�t�@�N�^
 * @param[in] eps �ɘa�W��
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void pbfPositionCorrection(float4* ppos, float* pscl, float4* pdp, rxParticleCell cell)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	float3 pos = make_float3(FETCHC(dSortedPos, index));	// �p�[�e�B�N���ʒu
	//int3 grid_pos = calcGridPos(pos);	// �p�[�e�B�N����������O���b�h�ʒu

	float h = params.EffectiveRadius;

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos-make_float3(h));
	grid_pos1 = calcGridPos(pos+make_float3(h));

	// ���͂̃O���b�h���܂߂ċߖT�T���C�ʒu�C���ʂ��v�Z
	float3 dpij = make_float3(0.0f);
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				dpij += calPositionCorrectionCell(n_grid_pos, index, pos, pscl, cell);
			}
		}
	}

	// �p�[�e�B�N���̃\�[�g�Ȃ��z���ł̃C���f�b�N�X
	uint oIdx = cell.dSortedIndex[index];

	// �ʒu�C����
	pdp[oIdx] = make_float4(dpij, 0.0);
}

/*!
 * �p�[�e�B�N���ʒu�C��
 * @param[inout] pos �p�[�e�B�N���ʒu
 * @param[in] pdp �ʒu�C����
 * @param[in] nprts �p�[�e�B�N����
 */
__global__
void pbfCorrectPosition(float4* ppos, float4* pdp, uint nprts)
{
	uint index = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= nprts) return;

	// �ʒu�C��
	ppos[index] += pdp[index];
}

/*!
 * ���x�ϓ��̌v�Z
 * @param[inout] pos �p�[�e�B�N���ʒu
 * @param[in] pdp �ʒu�C����
 * @param[in] nprts �p�[�e�B�N����
 */
__global__
void pbfDensityFluctuation(float* perr, float* pdens, float rest_dens, uint nprts)
{
	uint index = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= nprts) return;

	// ���x�ϓ�
	//perr[index] = fabs(pdens[index]-rest_dens)/rest_dens;
	float err = pdens[index]-rest_dens;
	perr[index] = (err >= 0.0f ? err : 0.0f)/rest_dens;
}




/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋������疧�x���v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] index �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float calBoundaryDensityCellPB(int3 gridPos, uint i, float3 pos0, float* dVolB, rxParticleCell bcell)
{
	uint gridHash = calcGridHashB(gridPos, params.GridSizeB);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = bcell.dCellStart[gridHash];

	float h = params.EffectiveRadius;
	float dens = 0.0f;
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = bcell.dCellEnd[gridHash];
		for(uint j = startIndex; j < endIndex; ++j){
			//if(j == i) continue;

			float3 pos1 = make_float3(bcell.dSortedPos[j]);
			uint jdx = bcell.dSortedIndex[j];

			float3 rij = pos0-pos1;
			float r = length(rij);

			if(r <= h){
				float q = h*h-r*r;
				dens += params.Density*dVolB[jdx]*params.Wpoly6*q*q*q;
			}
		}
	}

	return dens;
}

/*!
 * �p�[�e�B�N�����x�v�Z(�J�[�l���֐�)
 * @param[out] newDens �p�[�e�B�N�����x
 * @param[out] newPres �p�[�e�B�N������
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void pbfCalBoundaryDensity(float* newDens, float4* dPos, float* dVolB, rxParticleCell bcell, uint pnum)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= pnum) return;	
	
	float3 pos = make_float3(dPos[index]);	// �p�[�e�B�N���ʒu
	float h = params.EffectiveRadius;

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPosB(pos-make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);
	grid_pos1 = calcGridPosB(pos+make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);

	// ���͂̃O���b�h���܂߂ċߖT�T���C���x�v�Z
	float dens = 0.0f;
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				dens += calBoundaryDensityCellPB(n_grid_pos, index, pos, dVolB, bcell);
			}
		}
	}

	// ���x�����ʂɏ�������
	newDens[index] += dens;
}




/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋�������X�P�[�����O�t�@�N�^�̕��ꍀ�v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] index �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float calBoundaryScalingFactorCell(int3 gridPos, uint i, float3 pos0, float* dVolB, rxParticleCell bcell)
{
	uint gridHash = calcGridHashB(gridPos, params.GridSizeB);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = bcell.dCellStart[gridHash];

	float h = params.EffectiveRadius;
	float r0 = params.Density;
	float sd = 0.0f;
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = bcell.dCellEnd[gridHash];
		for(uint j = startIndex; j < endIndex; ++j){
			float3 pos1 = make_float3(bcell.dSortedPos[j]);
			uint jdx = bcell.dSortedIndex[j];

			float3 rij = pos0-pos1;
			float r = length(rij);

			if(r <= h && r > 0.0){
				float q = h-r;

				// Spiky�J�[�l���ňʒu�ϓ����v�Z
				float3 dp = (params.Density*dVolB[jdx]/params.Mass)*(params.GWspiky*q*q*rij/r)/r0;

				sd += dot(dp, dp);
			}

		}
	}

	return sd;
}

/*!
 * �X�P�[�����O�t�@�N�^�̌v�Z(���E�p�[�e�B�N���܂�)
 * @param[in] ppos �p�[�e�B�N�����S���W
 * @param[out] pdens �p�[�e�B�N�����x
 * @param[out] pscl �X�P�[�����O�t�@�N�^
 * @param[in] eps �ɘa�W��
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void pbfCalScalingFactorWithBoundary(float4* ppos, float* pdens, float* pscl, float eps, rxParticleCell cell, 
										float* bvol, rxParticleCell bcell)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	float3 pos = make_float3(FETCHC(dSortedPos, index));	// �p�[�e�B�N���ʒu
	//int3 grid_pos = calcGridPos(pos);	// �p�[�e�B�N����������O���b�h�ʒu

	float h = params.EffectiveRadius;
	float r0 = params.Density;

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos-make_float3(h));
	grid_pos1 = calcGridPos(pos+make_float3(h));

	// ���̃p�[�e�B�N���ɂ�閧�x
	float dens = 0.0f;
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				dens += calDensityCellPB(n_grid_pos, index, pos, cell);
			}
		}
	}

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos2, grid_pos3;
	grid_pos2 = calcGridPosB(pos-make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);
	grid_pos3 = calcGridPosB(pos+make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);

	// ���E�p�[�e�B�N���ɂ�閧�x
	for(int z = grid_pos2.z; z <= grid_pos3.z; ++z){
		for(int y = grid_pos2.y; y <= grid_pos3.y; ++y){
			for(int x = grid_pos2.x; x <= grid_pos3.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				dens += calBoundaryDensityCellPB(n_grid_pos, index, pos, bvol, bcell);
			}
		}
	}

	// ���x�S������(��(1))
	float C = dens/r0-1.0;

	// ���̃p�[�e�B�N���ɂ��X�P�[�����O�t�@�N�^�̕��ꍀ�v�Z
	float sd = 0.0f;
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				sd += calScalingFactorCell(n_grid_pos, index, pos, cell);
			}
		}
	}

	// ���E�p�[�e�B�N���ɂ��X�P�[�����O�t�@�N�^�̕��ꍀ�v�Z
	for(int z = grid_pos2.z; z <= grid_pos3.z; ++z){
		for(int y = grid_pos2.y; y <= grid_pos3.y; ++y){
			for(int x = grid_pos2.x; x <= grid_pos3.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				sd += calBoundaryScalingFactorCell(n_grid_pos, index, pos, bvol, bcell);
			}
		}
	}

	// �p�[�e�B�N���̃\�[�g�Ȃ��z���ł̃C���f�b�N�X
	uint oIdx = cell.dSortedIndex[index];

	// �X�P�[�����O�t�@�N�^�̌v�Z(��(11))
	pscl[oIdx] = -C/(sd+eps);

	// �X�V���ꂽ���x
	pdens[oIdx] = dens;
}



/*!
 * �X�P�[�����O�t�@�N�^�̌v�Z(���E�p�[�e�B�N���܂�)
 * @param[in] ppos �p�[�e�B�N�����S���W
 * @param[out] pdens �p�[�e�B�N�����x
 * @param[out] pscl �X�P�[�����O�t�@�N�^
 * @param[in] eps �ɘa�W��
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void pbfCalBoundaryScalingFactor(float4* ppos, float* pdens, float eps, rxParticleCell cell, 
									float* bvol, float* bscl, rxParticleCell bcell)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= bcell.uNumParticles) return;	
	
	float3 pos = make_float3(bcell.dSortedPos[index]);	// �p�[�e�B�N���ʒu

	float h = params.EffectiveRadius;
	float r0 = params.Density;

	// �p�[�e�B�N�����͂̃O���b�h(���̃p�[�e�B�N���p)
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos-make_float3(h));
	grid_pos1 = calcGridPos(pos+make_float3(h));

	// ���̃p�[�e�B�N���ɂ�閧�x
	float dens = 0.0f;
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				dens += calDensityCellPB(n_grid_pos, index, pos, cell);
			}
		}
	}

	// �p�[�e�B�N�����͂̃O���b�h(���E�p�[�e�B�N���p)
	int3 grid_pos2, grid_pos3;
	grid_pos2 = calcGridPosB(pos-make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);
	grid_pos3 = calcGridPosB(pos+make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);

	// ���E�p�[�e�B�N���ɂ�閧�x
	for(int z = grid_pos2.z; z <= grid_pos3.z; ++z){
		for(int y = grid_pos2.y; y <= grid_pos3.y; ++y){
			for(int x = grid_pos2.x; x <= grid_pos3.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				dens += calBoundaryDensityCellPB(n_grid_pos, index, pos, bvol, bcell);
			}
		}
	}

	// ���x�S������(��(1))
	float C = dens/r0-1.0;

	// ���̃p�[�e�B�N���ɂ��X�P�[�����O�t�@�N�^�̕��ꍀ�v�Z
	float sd = 0.0f;
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				sd += calScalingFactorCell(n_grid_pos, index, pos, cell);
			}
		}
	}

	// ���E�p�[�e�B�N���ɂ��X�P�[�����O�t�@�N�^�̕��ꍀ�v�Z
	for(int z = grid_pos2.z; z <= grid_pos3.z; ++z){
		for(int y = grid_pos2.y; y <= grid_pos3.y; ++y){
			for(int x = grid_pos2.x; x <= grid_pos3.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				sd += calBoundaryScalingFactorCell(n_grid_pos, index, pos, bvol, bcell);
			}
		}
	}

	// �p�[�e�B�N���̃\�[�g�Ȃ��z���ł̃C���f�b�N�X
	uint oIdx = bcell.dSortedIndex[index];

	// �X�P�[�����O�t�@�N�^�̌v�Z(��(11))
	bscl[oIdx] = -C/(sd+eps);
}



/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋�������X�P�[�����O�t�@�N�^�̕��ꍀ�v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] index �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float3 calBoundaryPositionCorrectionCell(int3 gridPos, uint i, float3 pos0, float si, float* bscl, float* bvol, rxParticleCell bcell)
{
	uint gridHash = calcGridHashB(gridPos, params.GridSizeB);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = bcell.dCellStart[gridHash];

	float k = params.AP_K;
	float n = params.AP_N;
	float wq = params.AP_WQ;

	float h = params.EffectiveRadius;
	float r0 = params.Density;
	float3 dp = make_float3(0.0);

	float dt = params.Dt;

	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = bcell.dCellEnd[gridHash];
		for(uint j = startIndex; j < endIndex; ++j){
			float3 pos1 = make_float3(bcell.dSortedPos[j]);
			uint jdx = bcell.dSortedIndex[j];

			float3 rij = pos0-pos1;
			float r = length(rij);

			if(r <= h && r > 0.0){
				float scorr = 0.0f;

				if(params.AP){
					float q1 = h*h-r*r;
					float ww = (params.Density*bvol[jdx]/params.Mass)*params.Wpoly6*q1*q1*q1/wq;
					scorr = -k*pow(ww, n)*dt*dt;
				}
				float q = h-r;
				float sj = bscl[jdx];

				// Spiky�J�[�l���ňʒu�C���ʂ��v�Z
				dp += (si+sj+scorr)*(params.GWspiky*q*q*rij/r)/r0;
			}

		}
	}

	return dp;
}

/*!
 * �X�P�[�����O�t�@�N�^�̌v�Z
 * @param[in] ppos �p�[�e�B�N�����S���W
 * @param[out] pdens �p�[�e�B�N�����x
 * @param[out] pscl �X�P�[�����O�t�@�N�^
 * @param[in] eps �ɘa�W��
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void pbfPositionCorrectionWithBoundary(float4* ppos, float* pscl, float4* pdp, rxParticleCell cell, 
										  float* bvol, float* bscl, rxParticleCell bcell)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	float3 pos = make_float3(FETCHC(dSortedPos, index));	// �p�[�e�B�N���ʒu
	//int3 grid_pos = calcGridPos(pos);	// �p�[�e�B�N����������O���b�h�ʒu

	float h = params.EffectiveRadius;

	float si = pscl[cell.dSortedIndex[index]];


	// �p�[�e�B�N�����͂̃O���b�h(���̃p�[�e�B�N���p)
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos-make_float3(h));
	grid_pos1 = calcGridPos(pos+make_float3(h));

	// ���̃p�[�e�B�N���ɂ��ʒu�C���ʂ��v�Z
	float3 dpij = make_float3(0.0f);
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				dpij += calPositionCorrectionCell(n_grid_pos, index, pos, pscl, cell);
			}
		}
	}

	// �p�[�e�B�N�����͂̃O���b�h(���E�p�[�e�B�N���p)
	int3 grid_pos2, grid_pos3;
	grid_pos2 = calcGridPosB(pos-make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);
	grid_pos3 = calcGridPosB(pos+make_float3(h), params.WorldOriginB, params.CellWidthB, params.GridSizeB);

	// ���E�p�[�e�B�N���ɂ��ʒu�C���ʂ��v�Z
	for(int z = grid_pos2.z; z <= grid_pos3.z; ++z){
		for(int y = grid_pos2.y; y <= grid_pos3.y; ++y){
			for(int x = grid_pos2.x; x <= grid_pos3.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				dpij += calBoundaryPositionCorrectionCell(n_grid_pos, index, pos, si, bscl, bvol, bcell);
			}
		}
	}

	// �p�[�e�B�N���̃\�[�g�Ȃ��z���ł̃C���f�b�N�X
	uint oIdx = cell.dSortedIndex[index];

	// �ʒu�C����
	pdp[oIdx] = make_float4(dpij, 0.0);
}


__device__
void calCollisionSolidPB(float3 &pos, float3 &vel, float dt)
{
	float d;
	float3 n;
	float3 cp;

	// �{�b�N�X�`��̃I�u�W�F�N�g�Ƃ̏Փ�
#if MAX_BOX_NUM
	for(int i = 0; i < params.BoxNum; ++i){
		if(params.BoxFlg[i] == 0) continue;
		
		collisionPointBox(pos, params.BoxCen[i], params.BoxExt[i], params.BoxRot[i], params.BoxInvRot[i], cp, d, n);

		if(d < 0.0){
			float res = params.Restitution;
			res = (res > 0) ? (res*fabs(d)/(dt*length(vel))) : 0.0f;
			vel -= (1+res)*n*dot(n, vel);
			pos = cp;
		}
	}
#endif

	// ���`��̃I�u�W�F�N�g�Ƃ̏Փ�
#if MAX_SPHERE_NUM
	for(int i = 0; i < params.SphereNum; ++i){
		if(params.SphereFlg[i] == 0) continue;

		collisionPointSphere(pos, params.SphereCen[i], params.SphereRad[i], cp, d, n);

		if(d < 0.0){
			float res = params.Restitution;
			res = (res > 0) ? (res*fabs(d)/(dt*length(vel))) : 0.0f;
			vel -= (1+res)*n*dot(n, vel);
			pos = cp;
		}
	}
#endif

	// ���͂̋��E�Ƃ̏Փ˔���
	float3 l0 = params.Boundary[0];
	float3 l1 = params.Boundary[1];
	collisionPointAABB(pos, 0.5*(l1+l0), 0.5*(l1-l0), cp, d, n);

	if(d < 0.0){
		float res = params.Restitution;
		res = (res > 0) ? (res*fabs(d)/(dt*length(vel))) : 0.0f;
		vel -= (1+res)*n*dot(n, vel);
		pos = cp;
	}
}

__device__
inline bool calCollisionPolygonPB(float3 &pos0, float3 &pos1, float3 &vel, float3 v0, float3 v1, float3 v2, float dt)
{
	float3 cp, n;
	if(intersectSegmentTriangle(pos0, pos1, v0, v1, v2, cp, n, params.ParticleRadius) == 1){
		float d = length(pos1-cp);
		n = normalize(n);

		//float res = params.Restitution;
		//res = (res > 0) ? (res*fabs(d)/(dt*length(vel))) : 0.0f;
		//float3 vr = -(1+res)*n*dot(n, vel);

		float3 v = pos1-pos0;
		float l = length(v);
		v /= l;
		float3 vd = v*(l-d);
		float3 vr = vd-2*dot(n, vd)*n;

		pos1 = cp+vr*0.7;

		//vel += vr;//+params.PolyVel[0];
		//vel.x = 1.0;

		return true;
	}
	return false;
}




/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V
 * @param[inout] ppos �p�[�e�B�N���ʒu
 * @param[inout] pvel �p�[�e�B�N�����x
 * @param[in] pfrc �p�[�e�B�N���ɂ������
 * @param[in] dens �p�[�e�B�N�����x
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
__global__
void pbfIntegrate(float4* ppos, float4* pvel, float4* pacc, 
					 float4* new_ppos, float4* new_pvel, float dt, uint nprts)
{
	uint index = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= nprts) return;

	float3 x = make_float3(ppos[index]);
	float3 v = make_float3(pvel[index]);
	float3 a = make_float3(pacc[index]);
	//float3 v_old = v;

	// �X�V�ʒu�C���x�̍X�V
	v += dt*a;
	x += dt*v;

	// �ő́E���E�Ƃ̏Փ�
	calCollisionSolidPB(x, v, dt);

	// �ʒu�Ƒ��x�̍X�V
	new_ppos[index] = make_float4(x);
	new_pvel[index] = make_float4(v);
}



/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V(Leap-Frog)
 * @param[inout] ppos �p�[�e�B�N���ʒu
 * @param[inout] pvel �p�[�e�B�N�����x
 * @param[in] pfrc �p�[�e�B�N���ɂ������
 * @param[in] dens �p�[�e�B�N�����x
 * @param[in] vrts
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
__global__
void pbfIntegrateWithPolygon(float4* ppos, float4* pvel, float4* pacc, 
								float4* new_ppos, float4* new_pvel, 
								float3* vrts, int3* tris, int tri_num, float dt, rxParticleCell cell)
{
	uint index = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;

	float3 x = make_float3(ppos[index]);
	float3 v = make_float3(pvel[index]);
	float3 a = make_float3(pacc[index]);
	//float3 v_old = v;
	float3 x_old = x;

	// �X�V�ʒu�C���x�̍X�V
	v += dt*a;
	x += dt*v;

	// �|���S���I�u�W�F�N�g�Ƃ̏Փ�
	int3 gridPos[2];
	gridPos[0] = calcGridPos(x_old);	// �ʒu�X�V�O�̃p�[�e�B�N����������O���b�h
	gridPos[1] = calcGridPos(x);		// �ʒu�X�V��̃p�[�e�B�N����������O���b�h
	for(int i = 0; i < 2; ++i){
		uint grid_hash = calcGridHash(gridPos[i]);
		uint start_index = cell.dPolyCellStart[grid_hash];
		if(start_index != 0xffffffff){	// �Z������łȂ����̃`�F�b�N

			uint end_index = cell.dPolyCellEnd[grid_hash];
			for(uint j = start_index; j < end_index; ++j){
				uint pidx = cell.dSortedPolyIdx[j];

				int3 idx = tris[pidx];
				if(calCollisionPolygonPB(x_old, x, v, vrts[idx.x], vrts[idx.y], vrts[idx.z], dt)){
				}
			}
		}
	}

	// �ő́E���E�Ƃ̏Փ�
	calCollisionSolidPB(x, v, dt);

	// �ʒu�Ƒ��x�̍X�V
	new_ppos[index] = make_float4(x);
	new_pvel[index] = make_float4(v);
}



/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V
 * @param[inout] ppos �p�[�e�B�N���ʒu
 * @param[inout] pvel �p�[�e�B�N�����x
 * @param[in] pfrc �p�[�e�B�N���ɂ������
 * @param[in] dens �p�[�e�B�N�����x
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
__global__
void pbfIntegrate2(float4* ppos, float4* pvel, float4* pacc, 
					  float4* new_ppos, float4* new_pvel, float dt, uint nprts)
{
	uint index = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= nprts) return;

	float3 x = make_float3(new_ppos[index]);
	float3 v = make_float3(new_pvel[index]);

	// �ő́E���E�Ƃ̏Փ�
	calCollisionSolidPB(x, v, dt);

	// �ʒu�Ƒ��x�̍X�V
	new_ppos[index] = make_float4(x);
}



/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V(Leap-Frog)
 * @param[inout] ppos �p�[�e�B�N���ʒu
 * @param[inout] pvel �p�[�e�B�N�����x
 * @param[in] pfrc �p�[�e�B�N���ɂ������
 * @param[in] dens �p�[�e�B�N�����x
 * @param[in] vrts
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
__global__
void pbfIntegrateWithPolygon2(float4* ppos, float4* pvel, float4* pacc, 
							  float4* new_ppos, float4* new_pvel, 
							  float3* vrts, int3* tris, int tri_num, float dt, rxParticleCell cell)
{
	uint index = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;

	float3 x = make_float3(new_ppos[index]);
	float3 x_old = make_float3(ppos[index]);
	float3 v = make_float3(new_pvel[index]);

	// �|���S���I�u�W�F�N�g�Ƃ̏Փ�
	int3 gridPos[2];
	gridPos[0] = calcGridPos(x_old);	// �ʒu�X�V�O�̃p�[�e�B�N����������O���b�h
	gridPos[1] = calcGridPos(x);		// �ʒu�X�V��̃p�[�e�B�N����������O���b�h
	for(int i = 0; i < 2; ++i){
		uint grid_hash = calcGridHash(gridPos[i]);
		uint start_index = cell.dPolyCellStart[grid_hash];
		if(start_index != 0xffffffff){	// �Z������łȂ����̃`�F�b�N

			uint end_index = cell.dPolyCellEnd[grid_hash];
			for(uint j = start_index; j < end_index; ++j){
				uint pidx = cell.dSortedPolyIdx[j];

				int3 idx = tris[pidx];
				if(calCollisionPolygonPB(x_old, x, v, vrts[idx.x], vrts[idx.y], vrts[idx.z], dt)){
				}
			}
		}
	}

	// �ő́E���E�Ƃ̏Փ�
	calCollisionSolidPB(x, v, dt);

	// �ʒu�Ƒ��x�̍X�V
	new_ppos[index] = make_float4(x);
	new_pvel[index] = make_float4(v);
}


/*!
 * �p�[�e�B�N���ʒu�C���x�̍X�V
 * @param[in] ppos �X�V���ꂽ�p�[�e�B�N���ʒu
 * @param[inout] new_ppos �X�e�b�v�ŏ��̃p�[�e�B�N���ʒu/�V�����p�[�e�B�N�����x
 * @param[out] new_pvel �V�����p�[�e�B�N�����x
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
__global__
void pbfUpdatePosition(float4* ppos, float4* new_ppos, float4* new_pvel, float dt, uint nprts)
{
	uint index = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= nprts) return;

	float3 x0 = make_float3(new_ppos[index]);
	float3 x1 = make_float3(ppos[index]);
	float3 v = (x1-x0)/dt;

	// �ʒu�Ƒ��x�̍X�V
	new_pvel[index] = make_float4(v);
	new_ppos[index] = make_float4(x1);
}

/*!
 * �p�[�e�B�N�����x�̍X�V
 * @param[in] ppos �X�V���ꂽ�p�[�e�B�N���ʒu
 * @param[in] new_ppos �X�e�b�v�ŏ��̃p�[�e�B�N���ʒu/�V�����p�[�e�B�N�����x
 * @param[out] new_pvel �V�����p�[�e�B�N�����x
 * @param[in] dt ���ԃX�e�b�v��
 * @param[in] nprts �p�[�e�B�N����
 */
__global__
void pbfUpdateVelocity(float4* ppos, float4* new_ppos, float4* new_pvel, float dt, uint nprts)
{
	uint index = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= nprts) return;

	float3 x0 = make_float3(new_ppos[index]);
	float3 x1 = make_float3(ppos[index]);
	float3 v = (x1-x0)/dt;

	// �ʒu�Ƒ��x�̍X�V
	new_pvel[index] = make_float4(v);
}



/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋������疧�x���v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] index �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float3 calXsphViscosityCell(int3 gridPos, uint i, float3 pos0, float3 vel0, float4* pvel, float* dens, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float h = params.EffectiveRadius;
	float3 v = make_float3(0.0);
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);
		for(uint j = startIndex; j < endIndex; ++j){
			//if(j == i) continue;

			float3 pos1 = make_float3(FETCHC(dSortedPos, j));

			float3 rij = pos0-pos1;
			float r = length(rij);

			if(r <= h){
				float3 vel1 = make_float3(pvel[cell.dSortedIndex[j]]);
				float3 rho1 = make_float3(dens[cell.dSortedIndex[j]]);

				float q = h*h-r*r;
				v += (params.Mass/rho1)*(vel1-vel0)*params.Wpoly6*q*q*q;
			}
		}
	}

	return v;
}

/*!
 * �p�[�e�B�N�����x�v�Z(�J�[�l���֐�)
 * @param[out] newDens �p�[�e�B�N�����x
 * @param[out] newPres �p�[�e�B�N������
 * @param[in]  cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void xsphVisocosity(float4* ppos, float4* pvel, float4* new_pvel, float* dens, float c, rxParticleCell cell)
{
	// �p�[�e�B�N���C���f�b�N�X
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	float3 pos0 = make_float3(FETCHC(dSortedPos, index));	// �p�[�e�B�N���ʒu
	float3 vel0 = make_float3(pvel[cell.dSortedIndex[index]]);	// �p�[�e�B�N�����x
	//int3 grid_pos = calcGridPos(pos0);	// �p�[�e�B�N����������O���b�h�ʒu
	float h = params.EffectiveRadius;

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos0-make_float3(h));
	grid_pos1 = calcGridPos(pos0+make_float3(h));

	// ���͂̃O���b�h���܂߂ċߖT�T���C���x�v�Z
	float3 v = make_float3(0.0);
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				v += calXsphViscosityCell(n_grid_pos, index, pos0, vel0, pvel, dens, cell);
			}
		}
	}

	// ���x�ƈ��͒l�����ʂɏ�������
	uint oIdx = cell.dSortedIndex[index];
	new_pvel[oIdx] = make_float4(vel0+c*v);
	//new_pvel[oIdx] = make_float4(vel0);
}




/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋������疧�x���v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] pos �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float calDensityCellGPB(int3 gridPos, float3 pos0, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float h = params.EffectiveRadius;
	float d = 0.0f;
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);

		for(uint j = startIndex; j < endIndex; ++j){
			//if(j != index){
				float3 pos1 = make_float3(FETCHC(dSortedPos, j));

				float3 rij = pos0-pos1;
				float r = length(rij);

				if(r <= h){
					float q = h*h-r*r;

					d += params.Mass*params.Wpoly6*q*q*q;
				}

			//}
		}
	}

	return d;
}

/*!
 * �O���b�h��ł̖��x���v�Z
 * @param[out] GridD �O���b�h���x
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] gnum �O���b�h��
 * @param[in] gmin �O���b�h�ŏ����W
 * @param[in] glen �O���b�h��
 */
__global__
void pbfCalDensityInGrid(float* GridD, rxParticleCell cell, 
					uint3 gnum, float3 gmin, float3 glen)
{
	uint blockId = __mul24(blockIdx.y, gridDim.x)+blockIdx.x;
	uint i = __mul24(blockId, blockDim.x)+threadIdx.x;

	uint3 gridPos = calcGridPosU(i, gnum);

	if(gridPos.x < gnum.x && gridPos.y < gnum.y && gridPos.z < gnum.z){
		float3 gpos;
		gpos.x = gmin.x+(gridPos.x)*glen.x;
		gpos.y = gmin.y+(gridPos.y)*glen.y;
		gpos.z = gmin.z+(gridPos.z)*glen.z;

		float d = 0.0f;

		int3 pgpos = calcGridPos(gpos);

		float h = params.EffectiveRadius;
		int3 grid_pos0, grid_pos1;
		grid_pos0 = calcGridPos(gpos-make_float3(h));
		grid_pos1 = calcGridPos(gpos+make_float3(h));

		for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
			for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
				for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
					int3 neighbourPos = make_int3(x, y, z);

					d += calDensityCellGPB(neighbourPos, gpos, cell);
				}
			}
		}

		GridD[gridPos.x+gridPos.y*gnum.x+gridPos.z*gnum.x*gnum.y] = d;
	}

}

/*!
 * �^����ꂽ�Z�����̃p�[�e�B�N���Ƃ̋�������@�����v�Z
 * @param[in] gridPos �O���b�h�ʒu
 * @param[in] i �p�[�e�B�N���C���f�b�N�X
 * @param[in] pos �v�Z���W
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @return �Z�����̃p�[�e�B�N������v�Z�������x�l
 */
__device__
float3 calNormalCellPB(int3 gridPos, uint i, float3 pos0, float* dens, rxParticleCell cell)
{
	uint gridHash = calcGridHash(gridPos);

	// �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
	uint startIndex = FETCHC(dCellStart, gridHash);

	float h = params.EffectiveRadius;
	float3 nrm = make_float3(0.0f);
	if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
		// �Z�����̃p�[�e�B�N���Ŕ���
		uint endIndex = FETCHC(dCellEnd, gridHash);

		for(uint j = startIndex; j < endIndex; ++j){
			if(j != i){
				float3 pos1 = make_float3(FETCHC(dSortedPos, j));

				float3 rij = pos0-pos1;
				float r = length(rij);

				if(r <= h && r > 0.0001){
					float d1 = dens[cell.dSortedIndex[j]];
					float q = h*h-r*r;

					nrm += (params.Mass/d1)*params.GWpoly6*q*q*rij;
				}

			}
		}
	}

	return nrm;
}


/*!
 * �p�[�e�B�N���@���v�Z(�J�[�l���֐�)
 * @param[out] newNrms �p�[�e�B�N���@��
 * @param[in] dens �p�[�e�B�N�����x
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 */
__global__
void pbfCalNormal(float4* newNrms, float* dens, rxParticleCell cell)
{
	uint index = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index >= cell.uNumParticles) return;	
	
	float3 pos = make_float3(FETCHC(dSortedPos, index));	// �p�[�e�B�N���ʒu
	float h = params.EffectiveRadius;
	//int3 grid_pos = calcGridPos(pos);	// �p�[�e�B�N����������O���b�h�ʒu

	// �p�[�e�B�N�����͂̃O���b�h
	int3 grid_pos0, grid_pos1;
	grid_pos0 = calcGridPos(pos-make_float3(h));
	grid_pos1 = calcGridPos(pos+make_float3(h));

	// ���͂̃O���b�h���܂߂ċߖT�T���C���x�v�Z
	float3 nrm = make_float3(0.0f);
	for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
		for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
			for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
				int3 n_grid_pos = make_int3(x, y, z);
				nrm += calNormalCellPB(n_grid_pos, index, pos, dens, cell);
			}
		}
	}

	float l = length(nrm);
	if(l > 0){
		nrm /= l;
	}

	uint oIdx = cell.dSortedIndex[index];
	newNrms[oIdx] = make_float4(nrm, 0.0f);
}





#endif // #ifndef _RX_PVSPH_KERNEL_CU_



